#include "hip/hip_runtime.h"
/*
 * -----------------------------------------------------------------
 * $Revision$
 * $Date$
 * ----------------------------------------------------------------- 
 * Programmer(s): Slaven Peles @ LLNL                               
 * -----------------------------------------------------------------
 * LLNS Copyright Start
 * Copyright (c) 2014, Lawrence Livermore National Security
 * This work was performed under the auspices of the U.S. Department 
 * of Energy by Lawrence Livermore National Laboratory in part under 
 * Contract W-7405-Eng-48 and in part under Contract DE-AC52-07NA27344.
 * Produced at the Lawrence Livermore National Laboratory.
 * All rights reserved.
 * For details, see the LICENSE file.
 * LLNS Copyright End
 * -----------------------------------------------------------------
 */

//#include <limits>

#include <nvector/raja/Vector.hpp>
#include <RAJA/RAJA.hxx>


extern "C" {

N_Vector N_VNewEmpty_Raja(long int length)
{
  N_Vector v;
  N_Vector_Ops ops;
  N_VectorContent_Raja content;

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);
  
  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvclone           = N_VClone_Raja;
  ops->nvcloneempty      = N_VCloneEmpty_Raja;
  ops->nvdestroy         = N_VDestroy_Raja;
  ops->nvspace           = N_VSpace_Raja;
  ops->nvgetarraypointer = NULL; //N_VGetArrayPointer_Raja;
  ops->nvsetarraypointer = NULL; //N_VSetArrayPointer_Raja;
  ops->nvlinearsum       = N_VLinearSum_Raja;
  ops->nvconst           = N_VConst_Raja;
  ops->nvprod            = N_VProd_Raja;
  ops->nvdiv             = N_VDiv_Raja;
  ops->nvscale           = N_VScale_Raja;
  ops->nvabs             = N_VAbs_Raja;
  ops->nvinv             = N_VInv_Raja;
  ops->nvaddconst        = N_VAddConst_Raja;
  ops->nvdotprod         = N_VDotProd_Raja;
  ops->nvmaxnorm         = N_VMaxNorm_Raja;
  ops->nvwrmsnormmask    = N_VWrmsNormMask_Raja;
  ops->nvwrmsnorm        = N_VWrmsNorm_Raja;
  ops->nvmin             = N_VMin_Raja;
//   ops->nvwl2norm         = N_VWL2Norm_Raja;
//   ops->nvl1norm          = N_VL1Norm_Raja;
//   ops->nvcompare         = N_VCompare_Raja;
//   ops->nvinvtest         = N_VInvTest_Raja;
//   ops->nvconstrmask      = N_VConstrMask_Raja;
//   ops->nvminquotient     = N_VMinQuotient_Raja;

  /* Create content */
  content = NULL;

  /* Attach content and ops */
  v->content = content;
  v->ops     = ops;

  return(v);
}

    
N_Vector N_VNew_Raja(long int length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Raja(length);
  if (v == NULL) return(NULL);

  v->content = new rvec::Vector<realtype, long int>(length);

  return(v);
}


N_Vector N_VMake_Raja(N_VectorContent_Raja c)
{
  N_Vector v;
  rvec::Vector<realtype, long int>* x = static_cast<rvec::Vector<realtype, long int>*>(c);
  long int length = x->size();

  v = NULL;
  v = N_VNewEmpty_Raja(length);
  if (v == NULL) return(NULL);

  v->content = c;

  return(v);
}


/* ----------------------------------------------------------------------------
 * Function to create an array of new RAJA-based vectors.
 */

N_Vector *N_VCloneVectorArray_Raja(int count, N_Vector w)
{
  N_Vector *vs;
  int j;

  if (count <= 0) return(NULL);

  vs = NULL;
  vs = (N_Vector *) malloc(count * sizeof(N_Vector));
  if(vs == NULL) return(NULL);

  for (j = 0; j < count; j++) {
    vs[j] = NULL;
    vs[j] = N_VClone_Raja(w);
    if (vs[j] == NULL) {
      N_VDestroyVectorArray_Raja(vs, j-1);
      return(NULL);
    }
  }

  return(vs);
}

/* ----------------------------------------------------------------------------
 * Function to create an array of new RAJA-based vectors with NULL data array.
 */

N_Vector *N_VCloneVectorArrayEmpty_Raja(int count, N_Vector w)
{
  N_Vector *vs;
  int j;

  if (count <= 0) return(NULL);

  vs = NULL;
  vs = (N_Vector *) malloc(count * sizeof(N_Vector));
  if(vs == NULL) return(NULL);

  for (j = 0; j < count; j++) {
    vs[j] = NULL;
    vs[j] = N_VCloneEmpty_Raja(w);
    if (vs[j] == NULL) {
      N_VDestroyVectorArray_Raja(vs, j-1);
      return(NULL);
    }
  }

  return(vs);
}

/* ----------------------------------------------------------------------------
 * Function to free an array created with N_VCloneVectorArray_Raja
 */

void N_VDestroyVectorArray_Raja(N_Vector *vs, int count)
{
  int j;

  for (j = 0; j < count; j++) N_VDestroy_Raja(vs[j]);

  free(vs); vs = NULL;

  return;
}



/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Raja(N_Vector w)
{
  N_Vector v;
  N_Vector_Ops ops;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvclone           = w->ops->nvclone;
  ops->nvcloneempty      = w->ops->nvcloneempty;
  ops->nvdestroy         = w->ops->nvdestroy;
  ops->nvspace           = w->ops->nvspace;
  ops->nvgetarraypointer = w->ops->nvgetarraypointer;
  ops->nvsetarraypointer = w->ops->nvsetarraypointer;
  ops->nvlinearsum       = w->ops->nvlinearsum;
  ops->nvconst           = w->ops->nvconst;
  ops->nvprod            = w->ops->nvprod;
  ops->nvdiv             = w->ops->nvdiv;
  ops->nvscale           = w->ops->nvscale;
  ops->nvabs             = w->ops->nvabs;
  ops->nvinv             = w->ops->nvinv;
  ops->nvaddconst        = w->ops->nvaddconst;
  ops->nvdotprod         = w->ops->nvdotprod;
  ops->nvmaxnorm         = w->ops->nvmaxnorm;
  ops->nvwrmsnormmask    = w->ops->nvwrmsnormmask;
  ops->nvwrmsnorm        = w->ops->nvwrmsnorm;
  ops->nvmin             = w->ops->nvmin;
  ops->nvwl2norm         = w->ops->nvwl2norm;
  ops->nvl1norm          = w->ops->nvl1norm;
  ops->nvcompare         = w->ops->nvcompare;
  ops->nvinvtest         = w->ops->nvinvtest;
  ops->nvconstrmask      = w->ops->nvconstrmask;
  ops->nvminquotient     = w->ops->nvminquotient;

  /* Create content */
  v->content = NULL;
  v->ops  = ops;

  return(v);
}

N_Vector N_VClone_Raja(N_Vector w)
{
  N_Vector v;
  rvec::Vector<double, long int>* wdat = static_cast<rvec::Vector<double, long int>*>(w->content);
  rvec::Vector<double, long int>* vdat = new rvec::Vector<double, long int>(*wdat);
  v = NULL;
  v = N_VCloneEmpty_Raja(w);
  if (v == NULL) return(NULL);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Raja(N_Vector v)
{
  rvec::Vector<double, long int>* x = static_cast<rvec::Vector<double, long int>*>(v->content);
  if (x != NULL) {
    if (!x->isClone()) {
      delete x;
      v->content = NULL;
    }
  }

  free(v->ops); v->ops = NULL;
  free(v); v = NULL;

  return;
}

void N_VSpace_Raja(N_Vector X, long int *lrw, long int *liw)
{
    *lrw = (extract_raja(X))->size();
    *liw = 1;
}

void N_VConst_Raja(realtype c, N_Vector Z)
{
  rvec::Vector<realtype, long int> *zv = extract_raja(Z);
  const long int N = zv->size();
  realtype *zdata = zv->device();

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = c;
  });
}

void N_VLinearSum_Raja(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  rvec::Vector<realtype, long int> *xv = extract_raja(X);
  rvec::Vector<realtype, long int> *yv = extract_raja(Y);
  rvec::Vector<realtype, long int> *zv = extract_raja(Z);
  const realtype *xdata = xv->device();
  const realtype *ydata = yv->device();
  const long int N = zv->size();
  realtype *zdata = zv->device();

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = a*xdata[i] + b*ydata[i];
  });
}

void N_VProd_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  rvec::Vector<realtype, long int> *xv = extract_raja(X);
  rvec::Vector<realtype, long int> *yv = extract_raja(Y);
  rvec::Vector<realtype, long int> *zv = extract_raja(Z);
  const realtype *xdata = xv->device();
  const realtype *ydata = yv->device();
  const long int N = zv->size();
  realtype *zdata = zv->device();

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = xdata[i] * ydata[i];
  });
}

void N_VDiv_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  rvec::Vector<realtype, long int> *xv = extract_raja(X);
  rvec::Vector<realtype, long int> *yv = extract_raja(Y);
  rvec::Vector<realtype, long int> *zv = extract_raja(Z);
  const realtype *xdata = xv->device();
  const realtype *ydata = yv->device();
  const long int N = zv->size();
  realtype *zdata = zv->device();

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = xdata[i] / ydata[i];
  });
}

void N_VScale_Raja(realtype c, N_Vector X, N_Vector Z)
{
  rvec::Vector<realtype, long int> *xv = extract_raja(X);
  rvec::Vector<realtype, long int> *zv = extract_raja(Z);
  const realtype *xdata = xv->device();
  const long int N = zv->size();
  realtype *zdata = zv->device();

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = c * xdata[i];
  });
}

void N_VAbs_Raja(N_Vector X, N_Vector Z)
{
  rvec::Vector<realtype, long int> *xv = extract_raja(X);
  rvec::Vector<realtype, long int> *zv = extract_raja(Z);
  const realtype *xdata = xv->device();
  const long int N = zv->size();
  realtype *zdata = zv->device();

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = std::abs(xdata[i]);
  });
}

void N_VInv_Raja(N_Vector X, N_Vector Z)
{
  rvec::Vector<realtype, long int> *xv = extract_raja(X);
  rvec::Vector<realtype, long int> *zv = extract_raja(Z);
  const realtype *xdata = xv->device();
  const long int N = zv->size();
  realtype *zdata = zv->device();

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = RCONST(1.0) / xdata[i];
  });
}

void N_VAddConst_Raja(N_Vector X, realtype b, N_Vector Z)
{
  rvec::Vector<realtype, long int> *xv = extract_raja(X);
  rvec::Vector<realtype, long int> *zv = extract_raja(Z);
  const realtype *xdata = xv->device();
  const long int N = zv->size();
  realtype *zdata = zv->device();

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = xdata[i] + b;
  });
}

realtype N_VDotProd_Raja(N_Vector X, N_Vector Y)
{
  rvec::Vector<realtype, long int>* xv = extract_raja(X);
  rvec::Vector<realtype, long int>* yv = extract_raja(Y);
  const realtype *xdata = xv->device();
  const realtype *ydata = yv->device();
  const long int N = xv->size();

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result += xdata[i] * ydata[i] ;
  });

  return static_cast<realtype>(gpu_result);
}

realtype N_VMaxNorm_Raja(N_Vector X)
{
  rvec::Vector<realtype, long int>* xv = extract_raja(X);
  const realtype *xdata = xv->device();
  const long int N = xv->size();

  RAJA::ReduceMax<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result.max(std::abs(xdata[i]));
  });

  return static_cast<realtype>(gpu_result);
}

realtype N_VWrmsNorm_Raja(N_Vector X, N_Vector W)
{
  rvec::Vector<realtype, long int>* xv = extract_raja(X);
  rvec::Vector<realtype, long int>* wv = extract_raja(W);
  const realtype *xdata = xv->device();
  const realtype *wdata = wv->device();
  const long int N = xv->size();

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
  });

  return std::sqrt(static_cast<realtype>(gpu_result)/N);
}

realtype N_VWrmsNormMask_Raja(N_Vector X, N_Vector W, N_Vector ID)
{
  rvec::Vector<realtype, long int>* xv = extract_raja(X);
  rvec::Vector<realtype, long int>* wv = extract_raja(W);
  rvec::Vector<realtype, long int>* idv = extract_raja(ID);
  const realtype *xdata = xv->device();
  const realtype *wdata = wv->device();
  const realtype *iddata = idv->device();
  const long int N = xv->size();

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i] * iddata[i]);
  });

  return std::sqrt(static_cast<realtype>(gpu_result)/N);
}

realtype N_VMin_Raja(N_Vector X)
{
  rvec::Vector<realtype, long int>* xv = extract_raja(X);
  const realtype *xdata = xv->device();
  const long int N = xv->size();

  RAJA::ReduceMin<RAJA::cuda_reduce<128>, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result.min(xdata[i]);
  });

  return static_cast<realtype>(gpu_result);
}


} // extern "C"
