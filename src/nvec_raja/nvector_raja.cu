#include "hip/hip_runtime.h"
/*
 * -----------------------------------------------------------------
 * $Revision$
 * $Date$
 * ----------------------------------------------------------------- 
 * Programmer(s): Slaven Peles @ LLNL                               
 * -----------------------------------------------------------------
 * LLNS Copyright Start
 * Copyright (c) 2014, Lawrence Livermore National Security
 * This work was performed under the auspices of the U.S. Department 
 * of Energy by Lawrence Livermore National Laboratory in part under 
 * Contract W-7405-Eng-48 and in part under Contract DE-AC52-07NA27344.
 * Produced at the Lawrence Livermore National Laboratory.
 * All rights reserved.
 * For details, see the LICENSE file.
 * LLNS Copyright End
 * -----------------------------------------------------------------
 */

//#include <limits>

#include <nvector/raja/Vector.hpp>
#include <RAJA/RAJA.hxx>

//#define abs(x) ((x)<0 ? -(x) : (x))

// Need better solution than defines
#define ZERO   RCONST(0.0)
#define HALF   RCONST(0.5)
#define ONE    RCONST(1.0)
#define ONEPT5 RCONST(1.5)

extern "C" {

/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Raja(N_Vector v)
{
  return SUNDIALS_NVEC_RAJA;
}

N_Vector N_VNewEmpty_Raja(long int length)
{
  N_Vector v;
  N_Vector_Ops ops;
  N_VectorContent_Raja content;

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);
  
  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = N_VGetVectorID_Raja;
  ops->nvclone           = N_VClone_Raja;
  ops->nvcloneempty      = N_VCloneEmpty_Raja;
  ops->nvdestroy         = N_VDestroy_Raja;
  ops->nvspace           = N_VSpace_Raja;
  ops->nvgetarraypointer = NULL; //N_VGetArrayPointer_Raja;
  ops->nvsetarraypointer = NULL; //N_VSetArrayPointer_Raja;
  ops->nvlinearsum       = N_VLinearSum_Raja;
  ops->nvconst           = N_VConst_Raja;
  ops->nvprod            = N_VProd_Raja;
  ops->nvdiv             = N_VDiv_Raja;
  ops->nvscale           = N_VScale_Raja;
  ops->nvabs             = N_VAbs_Raja;
  ops->nvinv             = N_VInv_Raja;
  ops->nvaddconst        = N_VAddConst_Raja;
  ops->nvdotprod         = N_VDotProd_Raja;
  ops->nvmaxnorm         = N_VMaxNorm_Raja;
  ops->nvwrmsnormmask    = N_VWrmsNormMask_Raja;
  ops->nvwrmsnorm        = N_VWrmsNorm_Raja;
  ops->nvmin             = N_VMin_Raja;
  ops->nvwl2norm         = N_VWL2Norm_Raja;
  ops->nvl1norm          = N_VL1Norm_Raja;
  ops->nvcompare         = N_VCompare_Raja;
  ops->nvinvtest         = N_VInvTest_Raja;
  ops->nvconstrmask      = N_VConstrMask_Raja;
  ops->nvminquotient     = N_VMinQuotient_Raja;

  /* Create content */
  content = NULL;

  /* Attach content and ops */
  v->content = content;
  v->ops     = ops;

  return(v);
}

    
N_Vector N_VNew_Raja(long int length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Raja(length);
  if (v == NULL) return(NULL);

  v->content = new sunrajavec::Vector<realtype, long int>(length);

  return(v);
}


N_Vector N_VMake_Raja(N_VectorContent_Raja c)
{
  N_Vector v;
  sunrajavec::Vector<realtype, long int>* x = static_cast<sunrajavec::Vector<realtype, long int>*>(c);
  long int length = x->size();

  v = NULL;
  v = N_VNewEmpty_Raja(length);
  if (v == NULL) return(NULL);

  v->content = c;

  return(v);
}


/* ----------------------------------------------------------------------------
 * Function to create an array of new RAJA-based vectors.
 */

N_Vector *N_VCloneVectorArray_Raja(int count, N_Vector w)
{
  N_Vector *vs;
  int j;

  if (count <= 0) return(NULL);

  vs = NULL;
  vs = (N_Vector *) malloc(count * sizeof(N_Vector));
  if(vs == NULL) return(NULL);

  for (j = 0; j < count; j++) {
    vs[j] = NULL;
    vs[j] = N_VClone_Raja(w);
    if (vs[j] == NULL) {
      N_VDestroyVectorArray_Raja(vs, j-1);
      return(NULL);
    }
  }

  return(vs);
}

/* ----------------------------------------------------------------------------
 * Function to create an array of new RAJA-based vectors with NULL data array.
 */

N_Vector *N_VCloneVectorArrayEmpty_Raja(int count, N_Vector w)
{
  N_Vector *vs;
  int j;

  if (count <= 0) return(NULL);

  vs = NULL;
  vs = (N_Vector *) malloc(count * sizeof(N_Vector));
  if(vs == NULL) return(NULL);

  for (j = 0; j < count; j++) {
    vs[j] = NULL;
    vs[j] = N_VCloneEmpty_Raja(w);
    if (vs[j] == NULL) {
      N_VDestroyVectorArray_Raja(vs, j-1);
      return(NULL);
    }
  }

  return(vs);
}

/* ----------------------------------------------------------------------------
 * Function to free an array created with N_VCloneVectorArray_Raja
 */

void N_VDestroyVectorArray_Raja(N_Vector *vs, int count)
{
  int j;

  for (j = 0; j < count; j++) N_VDestroy_Raja(vs[j]);

  free(vs); vs = NULL;

  return;
}


/* ----------------------------------------------------------------------------
 * Function to print the a serial vector
 */

void N_VPrint_Raja(N_Vector X)
{
  const realtype *xd = sunrajavec::getDevData<realtype, long int>(X);
  const long int N = sunrajavec::getSize<realtype, long int>(X);
  long int i;

  for (i = 0; i < N; ++i) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    printf("%35.32Lg\n", xd[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    printf("%19.16g\n", xd[i]);
#else
    printf("%11.8g\n", xd[i]);
#endif
  }
  printf("\n");

  return;
}

/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Raja(N_Vector w)
{
  N_Vector v;
  N_Vector_Ops ops;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = w->ops->nvgetvectorid;
  ops->nvclone           = w->ops->nvclone;
  ops->nvcloneempty      = w->ops->nvcloneempty;
  ops->nvdestroy         = w->ops->nvdestroy;
  ops->nvspace           = w->ops->nvspace;
  ops->nvgetarraypointer = w->ops->nvgetarraypointer;
  ops->nvsetarraypointer = w->ops->nvsetarraypointer;
  ops->nvlinearsum       = w->ops->nvlinearsum;
  ops->nvconst           = w->ops->nvconst;
  ops->nvprod            = w->ops->nvprod;
  ops->nvdiv             = w->ops->nvdiv;
  ops->nvscale           = w->ops->nvscale;
  ops->nvabs             = w->ops->nvabs;
  ops->nvinv             = w->ops->nvinv;
  ops->nvaddconst        = w->ops->nvaddconst;
  ops->nvdotprod         = w->ops->nvdotprod;
  ops->nvmaxnorm         = w->ops->nvmaxnorm;
  ops->nvwrmsnormmask    = w->ops->nvwrmsnormmask;
  ops->nvwrmsnorm        = w->ops->nvwrmsnorm;
  ops->nvmin             = w->ops->nvmin;
  ops->nvwl2norm         = w->ops->nvwl2norm;
  ops->nvl1norm          = w->ops->nvl1norm;
  ops->nvcompare         = w->ops->nvcompare;
  ops->nvinvtest         = w->ops->nvinvtest;
  ops->nvconstrmask      = w->ops->nvconstrmask;
  ops->nvminquotient     = w->ops->nvminquotient;

  /* Create content */
  v->content = NULL;
  v->ops  = ops;

  return(v);
}

N_Vector N_VClone_Raja(N_Vector w)
{
  N_Vector v;
  sunrajavec::Vector<realtype, long int>* wdat = static_cast<sunrajavec::Vector<realtype, long int>*>(w->content);
  sunrajavec::Vector<realtype, long int>* vdat = new sunrajavec::Vector<realtype, long int>(*wdat);
  v = NULL;
  v = N_VCloneEmpty_Raja(w);
  if (v == NULL) return(NULL);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Raja(N_Vector v)
{
  sunrajavec::Vector<realtype, long int>* x = static_cast<sunrajavec::Vector<realtype, long int>*>(v->content);
  if (x != NULL) {
    if (!x->isClone()) {
      delete x;
      v->content = NULL;
    }
  }

  free(v->ops); v->ops = NULL;
  free(v); v = NULL;

  return;
}

void N_VSpace_Raja(N_Vector X, long int *lrw, long int *liw)
{
    *lrw = sunrajavec::getSize<realtype, long int>(X);
    *liw = 1;
}

void N_VConst_Raja(realtype c, N_Vector Z)
{
  const long int N = sunrajavec::getSize<realtype, long int>(Z);
  realtype *zdata = sunrajavec::getDevData<realtype, long int>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = c;
  });
}

void N_VLinearSum_Raja(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const realtype *ydata = sunrajavec::getDevData<realtype, long int>(Y);
  const long int N = sunrajavec::getSize<realtype, long int>(X);
  realtype *zdata = sunrajavec::getDevData<realtype, long int>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = a*xdata[i] + b*ydata[i];
  });
}

void N_VProd_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const realtype *ydata = sunrajavec::getDevData<realtype, long int>(Y);
  const long int N = sunrajavec::getSize<realtype, long int>(X);
  realtype *zdata = sunrajavec::getDevData<realtype, long int>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = xdata[i] * ydata[i];
  });
}

void N_VDiv_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const realtype *ydata = sunrajavec::getDevData<realtype, long int>(Y);
  const long int N = sunrajavec::getSize<realtype, long int>(X);
  realtype *zdata = sunrajavec::getDevData<realtype, long int>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = xdata[i] / ydata[i];
  });
}

void N_VScale_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const long int N = sunrajavec::getSize<realtype, long int>(X);
  realtype *zdata = sunrajavec::getDevData<realtype, long int>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = c * xdata[i];
  });
}

void N_VAbs_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const long int N = sunrajavec::getSize<realtype, long int>(X);
  realtype *zdata = sunrajavec::getDevData<realtype, long int>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = abs(xdata[i]);
  });
}

void N_VInv_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const long int N = sunrajavec::getSize<realtype, long int>(X);
  realtype *zdata = sunrajavec::getDevData<realtype, long int>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = RCONST(1.0) / xdata[i];
  });
}

void N_VAddConst_Raja(N_Vector X, realtype b, N_Vector Z)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const long int N = sunrajavec::getSize<realtype, long int>(X);
  realtype *zdata = sunrajavec::getDevData<realtype, long int>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = xdata[i] + b;
  });
}

realtype N_VDotProd_Raja(N_Vector X, N_Vector Y)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const realtype *ydata = sunrajavec::getDevData<realtype, long int>(Y);
  const long int N = sunrajavec::getSize<realtype, long int>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result += xdata[i] * ydata[i] ;
  });

  return static_cast<realtype>(gpu_result);
}

realtype N_VMaxNorm_Raja(N_Vector X)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const long int N = sunrajavec::getSize<realtype, long int>(X);

  RAJA::ReduceMax<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result.max(abs(xdata[i]));
  });

  return static_cast<realtype>(gpu_result);
}

realtype N_VWrmsNorm_Raja(N_Vector X, N_Vector W)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const realtype *wdata = sunrajavec::getDevData<realtype, long int>(W);
  const long int N = sunrajavec::getSize<realtype, long int>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
  });

  return std::sqrt(static_cast<realtype>(gpu_result)/N);
}

realtype N_VWrmsNormMask_Raja(N_Vector X, N_Vector W, N_Vector ID)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const realtype *wdata = sunrajavec::getDevData<realtype, long int>(W);
  const realtype *iddata = sunrajavec::getDevData<realtype, long int>(ID);
  const long int N = sunrajavec::getSize<realtype, long int>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i] * iddata[i]);
  });

  return std::sqrt(static_cast<realtype>(gpu_result)/N);
}

realtype N_VMin_Raja(N_Vector X)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const long int N = sunrajavec::getSize<realtype, long int>(X);

  RAJA::ReduceMin<RAJA::cuda_reduce<128>, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result.min(xdata[i]);
  });

  return static_cast<realtype>(gpu_result);
}

realtype N_VWL2Norm_Raja(N_Vector X, N_Vector W)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const realtype *wdata = sunrajavec::getDevData<realtype, long int>(W);
  const long int N = sunrajavec::getSize<realtype, long int>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
  });

  return std::sqrt(static_cast<realtype>(gpu_result));
}

realtype N_VL1Norm_Raja(N_Vector X)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const long int N = sunrajavec::getSize<realtype, long int>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    gpu_result += (abs(xdata[i]));
  });

  return static_cast<realtype>(gpu_result);
}

void N_VCompare_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(X);
  const long int N = sunrajavec::getSize<realtype, long int>(X);
  realtype *zdata = sunrajavec::getDevData<realtype, long int>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(0, N, [=] __device__(long int i) {
     zdata[i] = abs(xdata[i]) >= c ? ONE : ZERO;
  });
}

booleantype N_VInvTest_Raja(N_Vector x, N_Vector z)
{
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(x);
  const long int N = sunrajavec::getSize<realtype, long int>(x);
  realtype *zdata = sunrajavec::getDevData<realtype, long int>(z);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(ZERO);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    if (xdata[i] == ZERO) {
      gpu_result += ONE;
    } else {
      zdata[i] = ONE/xdata[i];
    }
  });

  return (static_cast<realtype>(gpu_result) < HALF);
}

booleantype N_VConstrMask_Raja(N_Vector c, N_Vector x, N_Vector m)
{
  const realtype *cdata = sunrajavec::getDevData<realtype, long int>(c);
  const realtype *xdata = sunrajavec::getDevData<realtype, long int>(x);
  const long int N = sunrajavec::getSize<realtype, long int>(x);
  realtype *mdata = sunrajavec::getDevData<realtype, long int>(m);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(ZERO);
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    bool test = (abs(cdata[i]) > ONEPT5 && cdata[i]*xdata[i] <= ZERO) ||
                (abs(cdata[i]) > HALF   && cdata[i]*xdata[i] <  ZERO);
    mdata[i] = test ? ONE : ZERO;
    gpu_result += mdata[i];
  });

  return (static_cast<realtype>(gpu_result) < HALF);
}

realtype N_VMinQuotient_Raja(N_Vector num, N_Vector denom)
{
  const realtype *ndata = sunrajavec::getDevData<realtype, long int>(num);
  const realtype *ddata = sunrajavec::getDevData<realtype, long int>(denom);
  const long int N = sunrajavec::getSize<realtype, long int>(num);

  RAJA::ReduceMin<RAJA::cuda_reduce<128>, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall<RAJA::cuda_exec<128> >(0, N, [=] __device__(long int i) {
    if (ddata[i] != ZERO)
      gpu_result.min(ndata[i]/ddata[i]);
  });

  return (static_cast<realtype>(gpu_result));
}


} // extern "C"
