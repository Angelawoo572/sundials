#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * LLNS Copyright Start
 * Copyright (c) 2014, Lawrence Livermore National Security
 * This work was performed under the auspices of the U.S. Department
 * of Energy by Lawrence Livermore National Laboratory in part under
 * Contract W-7405-Eng-48 and in part under Contract DE-AC52-07NA27344.
 * Produced at the Lawrence Livermore National Laboratory.
 * All rights reserved.
 * For details, see the LICENSE file.
 * LLNS Copyright End
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <nvector/raja/Vector.hpp>
#include <RAJA/RAJA.hpp>
#include <sundials/sundials_mpi_types.h>


#define ZERO   RCONST(0.0)
#define HALF   RCONST(0.5)
#define ONE    RCONST(1.0)
#define ONEPT5 RCONST(1.5)

extern "C" {

using namespace sunrajavec;

static constexpr sunindextype zeroIdx = 0;
static realtype VAllReduce_Raja(realtype d, int op, SUNDIALS_Comm comm);

/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Raja(N_Vector v)
{
  return SUNDIALS_NVEC_RAJA;
}

N_Vector N_VNewEmpty_Raja(sunindextype length)
{
  N_Vector v;
  N_Vector_Ops ops;
  N_VectorContent_Raja content;

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = N_VGetVectorID_Raja;
  ops->nvclone           = N_VClone_Raja;
  ops->nvcloneempty      = N_VCloneEmpty_Raja;
  ops->nvdestroy         = N_VDestroy_Raja;
  ops->nvspace           = N_VSpace_Raja;
  ops->nvgetarraypointer = NULL; //N_VGetArrayPointer_Raja;
  ops->nvsetarraypointer = NULL; //N_VSetArrayPointer_Raja;
  ops->nvlinearsum       = N_VLinearSum_Raja;
  ops->nvconst           = N_VConst_Raja;
  ops->nvprod            = N_VProd_Raja;
  ops->nvdiv             = N_VDiv_Raja;
  ops->nvscale           = N_VScale_Raja;
  ops->nvabs             = N_VAbs_Raja;
  ops->nvinv             = N_VInv_Raja;
  ops->nvaddconst        = N_VAddConst_Raja;
  ops->nvdotprod         = N_VDotProd_Raja;
  ops->nvmaxnorm         = N_VMaxNorm_Raja;
  ops->nvwrmsnormmask    = N_VWrmsNormMask_Raja;
  ops->nvwrmsnorm        = N_VWrmsNorm_Raja;
  ops->nvmin             = N_VMin_Raja;
  ops->nvwl2norm         = N_VWL2Norm_Raja;
  ops->nvl1norm          = N_VL1Norm_Raja;
  ops->nvcompare         = N_VCompare_Raja;
  ops->nvinvtest         = N_VInvTest_Raja;
  ops->nvconstrmask      = N_VConstrMask_Raja;
  ops->nvminquotient     = N_VMinQuotient_Raja;

  /* Attach ops and set content to NULL */
  v->content = NULL;
  v->ops     = ops;

  return(v);
}


N_Vector N_VNew_Raja(SUNDIALS_Comm comm,
                     sunindextype local_length,
                     sunindextype global_length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Raja(local_length);
  if (v == NULL) return(NULL);

  v->content = new Vector<realtype, sunindextype>(comm, local_length, global_length);

  return(v);
}


N_Vector N_VMake_Raja(N_VectorContent_Raja c)
{
  N_Vector v;
  Vector<realtype, sunindextype>* x = static_cast<Vector<realtype, sunindextype>*>(c);
  sunindextype length = x->size();

  v = NULL;
  v = N_VNewEmpty_Raja(length);
  if (v == NULL) return(NULL);

  v->content = c;

  return(v);
}


/* -----------------------------------------------------------------
 * Function to return the length of the vector.
 */
sunindextype N_VGetLength_Raja(N_Vector v)
{
  Vector<realtype, sunindextype>* xd = static_cast<Vector<realtype, sunindextype>*>(v->content);
  return xd->size();
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw host data
 */

realtype *N_VGetHostArrayPointer_Raja(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  return (xv->host());
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw device data
 */

realtype *N_VGetDeviceArrayPointer_Raja(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  return (xv->device());
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Raja(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  xv->copyToDev();
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Raja(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  xv->copyFromDev();
}

/* ----------------------------------------------------------------------------
 * Function to print the a serial vector to stdout
 */

void N_VPrint_Raja(N_Vector X)
{
  N_VPrintFile_Raja(X, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a serial vector to outfile
 */

void N_VPrintFile_Raja(N_Vector X, FILE *outfile)
{
  const realtype *xd = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  sunindextype i;

  for (i = 0; i < N; ++i) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", xd[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", xd[i]);
#else
    fprintf(outfile, "%11.8g\n", xd[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}

/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Raja(N_Vector w)
{
  N_Vector v;
  N_Vector_Ops ops;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = w->ops->nvgetvectorid;
  ops->nvclone           = w->ops->nvclone;
  ops->nvcloneempty      = w->ops->nvcloneempty;
  ops->nvdestroy         = w->ops->nvdestroy;
  ops->nvspace           = w->ops->nvspace;
  ops->nvgetarraypointer = w->ops->nvgetarraypointer;
  ops->nvsetarraypointer = w->ops->nvsetarraypointer;
  ops->nvlinearsum       = w->ops->nvlinearsum;
  ops->nvconst           = w->ops->nvconst;
  ops->nvprod            = w->ops->nvprod;
  ops->nvdiv             = w->ops->nvdiv;
  ops->nvscale           = w->ops->nvscale;
  ops->nvabs             = w->ops->nvabs;
  ops->nvinv             = w->ops->nvinv;
  ops->nvaddconst        = w->ops->nvaddconst;
  ops->nvdotprod         = w->ops->nvdotprod;
  ops->nvmaxnorm         = w->ops->nvmaxnorm;
  ops->nvwrmsnormmask    = w->ops->nvwrmsnormmask;
  ops->nvwrmsnorm        = w->ops->nvwrmsnorm;
  ops->nvmin             = w->ops->nvmin;
  ops->nvwl2norm         = w->ops->nvwl2norm;
  ops->nvl1norm          = w->ops->nvl1norm;
  ops->nvcompare         = w->ops->nvcompare;
  ops->nvinvtest         = w->ops->nvinvtest;
  ops->nvconstrmask      = w->ops->nvconstrmask;
  ops->nvminquotient     = w->ops->nvminquotient;

  /* Create content */
  v->content = NULL;
  v->ops  = ops;

  return(v);
}

N_Vector N_VClone_Raja(N_Vector w)
{
  N_Vector v;
  Vector<realtype, sunindextype>* wdat = static_cast<Vector<realtype, sunindextype>*>(w->content);
  Vector<realtype, sunindextype>* vdat = new Vector<realtype, sunindextype>(*wdat);
  v = NULL;
  v = N_VCloneEmpty_Raja(w);
  if (v == NULL) return(NULL);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Raja(N_Vector v)
{
  Vector<realtype, sunindextype>* x = static_cast<Vector<realtype, sunindextype>*>(v->content);
  if (x != NULL) {
    delete x;
    v->content = NULL;
  }

  free(v->ops); v->ops = NULL;
  free(v); v = NULL;

  return;
}

void N_VSpace_Raja(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
    *lrw = getSize<realtype, sunindextype>(X);
    *liw = 1;
}

void N_VConst_Raja(realtype c, N_Vector Z)
{
  const sunindextype N = getSize<realtype, sunindextype>(Z);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = c;
  });
}

void N_VLinearSum_Raja(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *ydata = getDevData<realtype, sunindextype>(Y);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = a*xdata[i] + b*ydata[i];
  });
}

void N_VProd_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *ydata = getDevData<realtype, sunindextype>(Y);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = xdata[i] * ydata[i];
  });
}

void N_VDiv_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *ydata = getDevData<realtype, sunindextype>(Y);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = xdata[i] / ydata[i];
  });
}

void N_VScale_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = c * xdata[i];
  });
}

void N_VAbs_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = abs(xdata[i]);
  });
}

void N_VInv_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = RCONST(1.0) / xdata[i];
  });
}

void N_VAddConst_Raja(N_Vector X, realtype b, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = xdata[i] + b;
  });
}

realtype N_VDotProd_Raja(N_Vector X, N_Vector Y)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *ydata = getDevData<realtype, sunindextype>(Y);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result += xdata[i] * ydata[i] ;
  });

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  realtype gsum = VAllReduce_Raja(sum, 1, comm);
  return gsum;
}

realtype N_VMaxNorm_Raja(N_Vector X)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceMax<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result.max(abs(xdata[i]));
  });

  /* Reduce across MPI processes */
  realtype maximum = static_cast<realtype>(gpu_result);
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  return VAllReduce_Raja(maximum, 2, comm);
}

realtype N_VWrmsNorm_Raja(N_Vector X, N_Vector W)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *wdata = getDevData<realtype, sunindextype>(W);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  const sunindextype Nglobal = getGlobalSize<realtype, sunindextype>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
  });

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  return std::sqrt(VAllReduce_Raja(sum, 1, comm)/Nglobal);
}

realtype N_VWrmsNormMask_Raja(N_Vector X, N_Vector W, N_Vector ID)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *wdata = getDevData<realtype, sunindextype>(W);
  const realtype *iddata = getDevData<realtype, sunindextype>(ID);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  const sunindextype Nglobal = getGlobalSize<realtype, sunindextype>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i] * iddata[i]);
  });

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  return std::sqrt(VAllReduce_Raja(sum, 1, comm)/Nglobal);
}

realtype N_VMin_Raja(N_Vector X)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceMin<RAJA::cuda_reduce<128>, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result.min(xdata[i]);
  });

  /* Reduce across MPI processes */
  realtype minumum = static_cast<realtype>(gpu_result);
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  return VAllReduce_Raja(minumum, 3, comm);
}

realtype N_VWL2Norm_Raja(N_Vector X, N_Vector W)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *wdata = getDevData<realtype, sunindextype>(W);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
  });

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  return std::sqrt(VAllReduce_Raja(sum, 1, comm));
}

realtype N_VL1Norm_Raja(N_Vector X)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result += (abs(xdata[i]));
  });

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  return VAllReduce_Raja(sum, 1, comm);
}

void N_VCompare_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = abs(xdata[i]) >= c ? ONE : ZERO;
  });
}

booleantype N_VInvTest_Raja(N_Vector x, N_Vector z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(x);
  const sunindextype N = getSize<realtype, sunindextype>(x);
  realtype *zdata = getDevData<realtype, sunindextype>(z);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(ZERO);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    if (xdata[i] == ZERO) {
      gpu_result += ONE;
    } else {
      zdata[i] = ONE/xdata[i];
    }
  });

  /* Reduce across MPI processes */
  realtype minimum = static_cast<realtype>(gpu_result);
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(x);
  realtype global_minimum = VAllReduce_Raja(minimum, 3, comm);

  return (global_minimum < HALF);
}

booleantype N_VConstrMask_Raja(N_Vector c, N_Vector x, N_Vector m)
{
  const realtype *cdata = getDevData<realtype, sunindextype>(c);
  const realtype *xdata = getDevData<realtype, sunindextype>(x);
  const sunindextype N = getSize<realtype, sunindextype>(x);
  realtype *mdata = getDevData<realtype, sunindextype>(m);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(ZERO);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    bool test = (abs(cdata[i]) > ONEPT5 && cdata[i]*xdata[i] <= ZERO) ||
                (abs(cdata[i]) > HALF   && cdata[i]*xdata[i] <  ZERO);
    mdata[i] = test ? ONE : ZERO;
    gpu_result += mdata[i];
  });

  /* Reduce across MPI processes */
  realtype minimum = static_cast<realtype>(gpu_result);
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(x);
  realtype global_minimum = VAllReduce_Raja(minimum, 3, comm);

  return (global_minimum < HALF);
}

realtype N_VMinQuotient_Raja(N_Vector num, N_Vector denom)
{
  const realtype *ndata = getDevData<realtype, sunindextype>(num);
  const realtype *ddata = getDevData<realtype, sunindextype>(denom);
  const sunindextype N = getSize<realtype, sunindextype>(num);

  RAJA::ReduceMin<RAJA::cuda_reduce<128>, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    if (ddata[i] != ZERO)
      gpu_result.min(ndata[i]/ddata[i]);
  });

  /* Reduce across MPI processes */
  realtype minimum = static_cast<realtype>(gpu_result);
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(num);
  return VAllReduce_Raja(minimum, 3, comm);
}


/*
 * -----------------------------------------------------------------
 * private functions
 * -----------------------------------------------------------------
 */

static realtype VAllReduce_Raja(realtype d, int op, SUNDIALS_Comm comm)
{
  /*
   * This function does a global reduction.  The operation is
   *   sum if op = 1,
   *   max if op = 2,
   *   min if op = 3.
   * The operation is over all processors in the communicator
   */

#ifdef SUNDIALS_MPI_ENABLED

  realtype out;

  switch (op) {
   case 1: MPI_Allreduce(&d, &out, 1, PVEC_REAL_MPI_TYPE, MPI_SUM, comm);
           break;

   case 2: MPI_Allreduce(&d, &out, 1, PVEC_REAL_MPI_TYPE, MPI_MAX, comm);
           break;

   case 3: MPI_Allreduce(&d, &out, 1, PVEC_REAL_MPI_TYPE, MPI_MIN, comm);
           break;

   default: break;
  }

  return(out);

#else

  /* If MPI is not enabled don't do reduction */
  return d;

#endif // ifdef SUNDIALS_MPI_ENABLED
}


} // extern "C"
