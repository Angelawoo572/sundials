#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * LLNS Copyright Start
 * Copyright (c) 2014, Lawrence Livermore National Security
 * This work was performed under the auspices of the U.S. Department
 * of Energy by Lawrence Livermore National Laboratory in part under
 * Contract W-7405-Eng-48 and in part under Contract DE-AC52-07NA27344.
 * Produced at the Lawrence Livermore National Laboratory.
 * All rights reserved.
 * For details, see the LICENSE file.
 * LLNS Copyright End
 * -----------------------------------------------------------------
 * This is the implementation file for a serial implementation
 * of the NVECTOR package.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <nvector/cuda/Vector.hpp>
#include <nvector/cuda/VectorKernels.cuh>
#include <nvector/cuda/VectorArrayKernels.cuh>
#include <sundials/sundials_mpi.h>

#define ZERO   RCONST(0.0)
#define HALF   RCONST(0.5)
#define ONE    RCONST(1.0)
#define ONEPT5 RCONST(1.5)

extern "C" {

using namespace suncudavec;

/*
 * Type definitions
 */


/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Cuda(N_Vector v)
{
  return SUNDIALS_NVEC_CUDA;
}

N_Vector N_VNewEmpty_Cuda(sunindextype length)
{
  N_Vector v;
  N_Vector_Ops ops;

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = N_VGetVectorID_Cuda;
  ops->nvclone           = N_VClone_Cuda;
  ops->nvcloneempty      = N_VCloneEmpty_Cuda;
  ops->nvdestroy         = N_VDestroy_Cuda;
  ops->nvspace           = N_VSpace_Cuda;
  ops->nvgetarraypointer = NULL;
  ops->nvsetarraypointer = NULL;

  /* standard vector operations */
  ops->nvlinearsum    = N_VLinearSum_Cuda;
  ops->nvconst        = N_VConst_Cuda;
  ops->nvprod         = N_VProd_Cuda;
  ops->nvdiv          = N_VDiv_Cuda;
  ops->nvscale        = N_VScale_Cuda;
  ops->nvabs          = N_VAbs_Cuda;
  ops->nvinv          = N_VInv_Cuda;
  ops->nvaddconst     = N_VAddConst_Cuda;
  ops->nvdotprod      = N_VDotProd_Cuda;
  ops->nvmaxnorm      = N_VMaxNorm_Cuda;
  ops->nvwrmsnormmask = N_VWrmsNormMask_Cuda;
  ops->nvwrmsnorm     = N_VWrmsNorm_Cuda;
  ops->nvmin          = N_VMin_Cuda;
  ops->nvwl2norm      = N_VWL2Norm_Cuda;
  ops->nvl1norm       = N_VL1Norm_Cuda;
  ops->nvcompare      = N_VCompare_Cuda;
  ops->nvinvtest      = N_VInvTest_Cuda;
  ops->nvconstrmask   = N_VConstrMask_Cuda;
  ops->nvminquotient  = N_VMinQuotient_Cuda;

  /* fused vector operations */
  ops->nvlinearcombination = N_VLinearCombination_Cuda;
  ops->nvscaleaddmulti     = N_VScaleAddMulti_Cuda;
  ops->nvdotprodmulti      = NULL; // N_VDotProdMulti_Cuda;

  /* vector array operations */
  ops->nvlinearsumvectorarray         = N_VLinearSumVectorArray_Cuda;
  ops->nvscalevectorarray             = N_VScaleVectorArray_Cuda;
  ops->nvconstvectorarray             = N_VConstVectorArray_Cuda;
  ops->nvwrmsnormvectorarray          = N_VWrmsNormVectorArray_Cuda;
  ops->nvwrmsnormmaskvectorarray      = N_VWrmsNormMaskVectorArray_Cuda;
  ops->nvscaleaddmultivectorarray     = N_VScaleAddMultiVectorArray_Cuda;
  ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Cuda;

  /* Attach ops and set content to NULL */
  v->content = NULL;
  v->ops     = ops;

  return(v);
}


N_Vector N_VNew_Cuda(SUNDIALS_Comm comm,
                     sunindextype local_length,
                     sunindextype global_length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda(local_length);
  if (v == NULL)
    return(NULL);

  v->content = new Vector<realtype, sunindextype>(comm, local_length, global_length);

  return(v);
}


N_Vector N_VMake_Cuda(N_VectorContent_Cuda c)
{
  N_Vector v;
  Vector<realtype, sunindextype>* x = static_cast<Vector<realtype, sunindextype>*>(c);
  sunindextype length = x->size();

  v = NULL;
  v = N_VNewEmpty_Cuda(length);
  if (v == NULL) return(NULL);

  v->content = c;

  return(v);
}

/* -----------------------------------------------------------------
 * Function to return the length of the vector.
 */
sunindextype N_VGetLength_Cuda(N_Vector v)
{
  Vector<realtype, sunindextype>* xd = static_cast<Vector<realtype, sunindextype>*>(v->content);
  return xd->size();
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw host data
 */

realtype *N_VGetHostArrayPointer_Cuda(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  return (xv->host());
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw device data
 */

realtype *N_VGetDeviceArrayPointer_Cuda(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  return (xv->device());
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Cuda(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  xv->copyToDev();
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Cuda(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  xv->copyFromDev();
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to stdout
 */

void N_VPrint_Cuda(N_Vector x)
{
  N_VPrintFile_Cuda(x, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to outfile
 */

void N_VPrintFile_Cuda(N_Vector x, FILE *outfile)
{
  sunindextype i;
  Vector<realtype, sunindextype>* xd = static_cast<Vector<realtype, sunindextype>*>(x->content);

  for (i = 0; i < xd->size(); i++) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", xd->host()[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", xd->host()[i]);
#else
    fprintf(outfile, "%11.8g\n", xd->host()[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}


/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Cuda(N_Vector w)
{
  N_Vector v;
  N_Vector_Ops ops;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = w->ops->nvgetvectorid;
  ops->nvclone           = w->ops->nvclone;
  ops->nvcloneempty      = w->ops->nvcloneempty;
  ops->nvdestroy         = w->ops->nvdestroy;
  ops->nvspace           = w->ops->nvspace;
  ops->nvgetarraypointer = w->ops->nvgetarraypointer;
  ops->nvsetarraypointer = w->ops->nvsetarraypointer;

  /* standard vector operations */
  ops->nvlinearsum    = w->ops->nvlinearsum;
  ops->nvconst        = w->ops->nvconst;
  ops->nvprod         = w->ops->nvprod;
  ops->nvdiv          = w->ops->nvdiv;
  ops->nvscale        = w->ops->nvscale;
  ops->nvabs          = w->ops->nvabs;
  ops->nvinv          = w->ops->nvinv;
  ops->nvaddconst     = w->ops->nvaddconst;
  ops->nvdotprod      = w->ops->nvdotprod;
  ops->nvmaxnorm      = w->ops->nvmaxnorm;
  ops->nvwrmsnormmask = w->ops->nvwrmsnormmask;
  ops->nvwrmsnorm     = w->ops->nvwrmsnorm;
  ops->nvmin          = w->ops->nvmin;
  ops->nvwl2norm      = w->ops->nvwl2norm;
  ops->nvl1norm       = w->ops->nvl1norm;
  ops->nvcompare      = w->ops->nvcompare;
  ops->nvinvtest      = w->ops->nvinvtest;
  ops->nvconstrmask   = w->ops->nvconstrmask;
  ops->nvminquotient  = w->ops->nvminquotient;

  /* fused vector operations */
  ops->nvlinearcombination = w->ops->nvlinearcombination;
  ops->nvscaleaddmulti     = w->ops->nvscaleaddmulti;
  ops->nvdotprodmulti      = w->ops->nvdotprodmulti;

  /* vector array operations */
  ops->nvlinearsumvectorarray         = w->ops->nvlinearsumvectorarray;
  ops->nvscalevectorarray             = w->ops->nvscalevectorarray;
  ops->nvconstvectorarray             = w->ops->nvconstvectorarray;
  ops->nvwrmsnormvectorarray          = w->ops->nvwrmsnormvectorarray;
  ops->nvwrmsnormmaskvectorarray      = w->ops->nvwrmsnormmaskvectorarray;
  ops->nvscaleaddmultivectorarray     = w->ops->nvscaleaddmultivectorarray;
  ops->nvlinearcombinationvectorarray = w->ops->nvlinearcombinationvectorarray;

  /* Create content */
  v->content = NULL;
  v->ops  = ops;

  return(v);
}

N_Vector N_VClone_Cuda(N_Vector w)
{
  N_Vector v;
  Vector<realtype, sunindextype>* wdat = static_cast<Vector<realtype, sunindextype>*>(w->content);
  Vector<realtype, sunindextype>* vdat = new Vector<realtype, sunindextype>(*wdat);
  v = NULL;
  v = N_VCloneEmpty_Cuda(w);
  if (v == NULL) return(NULL);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Cuda(N_Vector v)
{
  Vector<realtype, sunindextype>* x = static_cast<Vector<realtype, sunindextype>*>(v->content);
  if (x != NULL) {
    delete x;
    v->content = NULL;
  }

  free(v->ops); v->ops = NULL;
  free(v); v = NULL;

  return;
}

void N_VSpace_Cuda(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  int npes;

  SUNDIALS_Comm_size(comm, &npes);

  *lrw = getGlobalSize<realtype, sunindextype>(X);
  *liw = 2*npes;
}

void N_VConst_Cuda(realtype a, N_Vector X)
{
  auto xvec = extract<realtype, sunindextype>(X);
  setConst(a, *xvec);
}

void N_VLinearSum_Cuda(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto yvec = extract<realtype, sunindextype>(Y);
  auto zvec = extract<realtype, sunindextype>(Z);
  linearSum(a, *xvec, b, *yvec, *zvec);
}

void N_VProd_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto yvec = extract<realtype, sunindextype>(Y);
  auto zvec = extract<realtype, sunindextype>(Z);
  prod(*xvec, *yvec, *zvec);
}

void N_VDiv_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto yvec = extract<realtype, sunindextype>(Y);
  auto zvec = extract<realtype, sunindextype>(Z);
  div(*xvec, *yvec, *zvec);
}

void N_VScale_Cuda(realtype a, N_Vector X, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  auto zvec = extract<realtype, sunindextype>(Z);
  scale(a, *xvec, *zvec);
}

void N_VAbs_Cuda(N_Vector X, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  auto zvec = extract<realtype, sunindextype>(Z);
  absVal(*xvec, *zvec);
}

void N_VInv_Cuda(N_Vector X, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  auto zvec = extract<realtype, sunindextype>(Z);
  inv(*xvec, *zvec);
}

void N_VAddConst_Cuda(N_Vector X, realtype b, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  auto zvec = extract<realtype, sunindextype>(Z);
  addConst(b, *xvec, *zvec);
}

realtype N_VDotProd_Cuda(N_Vector X, N_Vector Y)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto yvec = extract<realtype, sunindextype>(Y);

  realtype sum = dotProd(*xvec, *yvec);

  realtype gsum = SUNDIALS_Reduce(sum, 1, comm);
  return gsum;
}

realtype N_VMaxNorm_Cuda(N_Vector X)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);

  realtype locmax = maxNorm(*xvec);

  realtype globmax = SUNDIALS_Reduce(locmax, 2, comm);
  return globmax;
}

realtype N_VWrmsNorm_Cuda(N_Vector X, N_Vector W)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const sunindextype Nglob = getGlobalSize<realtype,sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto wvec = extract<realtype, sunindextype>(W);

  realtype sum = wL2NormSquare(*xvec, *wvec);

  realtype gsum = SUNDIALS_Reduce(sum, 1, comm);
  return std::sqrt(gsum/Nglob);
}

realtype N_VWrmsNormMask_Cuda(N_Vector X, N_Vector W, N_Vector Id)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const sunindextype Nglob = getGlobalSize<realtype,sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto wvec = extract<realtype, sunindextype>(W);
  const auto ivec = extract<realtype, sunindextype>(Id);

  realtype sum = wL2NormSquareMask(*xvec, *wvec, *ivec);

  realtype gsum = SUNDIALS_Reduce(sum, 1, comm);
  return std::sqrt(gsum/Nglob);
}

realtype N_VMin_Cuda(N_Vector X)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);

  realtype locmin = findMin(*xvec);

  realtype globmin = SUNDIALS_Reduce(locmin, 3, comm);
  return globmin;
}

realtype N_VWL2Norm_Cuda(N_Vector X, N_Vector W)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto wvec = extract<realtype, sunindextype>(W);

  realtype sum = wL2NormSquare(*xvec, *wvec);

  realtype gsum = SUNDIALS_Reduce(sum, 1, comm);
  return std::sqrt(gsum);
}

realtype N_VL1Norm_Cuda(N_Vector X)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);

  realtype sum = L1Norm(*xvec);

  realtype gsum = SUNDIALS_Reduce(sum, 1, comm);
  return gsum;
}

void N_VCompare_Cuda(realtype c, N_Vector X, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  auto zvec = extract<realtype, sunindextype>(Z);
  compare(c, *xvec, *zvec);
}

booleantype N_VInvTest_Cuda(N_Vector X, N_Vector Z)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto zvec = extract<realtype, sunindextype>(Z);

  realtype locmin = invTest(*xvec, *zvec);

  realtype globmin = SUNDIALS_Reduce(locmin, 3, comm);
  return (globmin < HALF);
}

/*
 * TODO:Fix bug in N_VConstrMask_Cuda kernel.
 * Currently an inefficient workaround is implemented.
 */
booleantype N_VConstrMask_Cuda(N_Vector C, N_Vector X, N_Vector M)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto cvec = extract<realtype, sunindextype>(C);
  const auto xvec = extract<realtype, sunindextype>(X);
  auto mvec = extract<realtype, sunindextype>(M);

  realtype locsum = constrMask(*cvec, *xvec, *mvec);

  realtype globsum = SUNDIALS_Reduce(locsum, 1, comm);
  return (globsum < HALF);
}

realtype N_VMinQuotient_Cuda(N_Vector num, N_Vector denom)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(num);
  const auto numvec = extract<realtype, sunindextype>(num);
  const auto denvec = extract<realtype, sunindextype>(denom);

  realtype locmin = minQuotient(*numvec, *denvec);

  realtype globmin = SUNDIALS_Reduce(locmin, 3, comm);
  return globmin;
}


/*
 * -----------------------------------------------------------------
 * fused vector operations
 * -----------------------------------------------------------------
 */

int N_VLinearCombination_Cuda(int nvec, realtype* c, N_Vector* X, N_Vector Z)
{
  hipError_t err;
  Vector<realtype, sunindextype>** Xv;
  Vector<realtype, sunindextype>*  Zv;

  Zv = extract<realtype, sunindextype>(Z);

  Xv = new Vector<realtype, sunindextype>*[nvec];
  for (int i=0; i<nvec; i++)
    Xv[i] = extract<realtype, sunindextype>(X[i]);

  err = linearCombination(nvec, c, Xv, Zv);

  delete[] Xv;

  if (err != hipSuccess)
    return(-1);
  else
    return(0);
}


int N_VScaleAddMulti_Cuda(int nvec, realtype* c, N_Vector X, N_Vector* Y,
                           N_Vector* Z)
{
  hipError_t err;
  Vector<realtype, sunindextype>*  Xv;
  Vector<realtype, sunindextype>** Yv;
  Vector<realtype, sunindextype>** Zv;

  Xv = extract<realtype, sunindextype>(X);

  Yv = new Vector<realtype, sunindextype>*[nvec];
  for (int i=0; i<nvec; i++)
    Yv[i] = extract<realtype, sunindextype>(Y[i]);

  Zv = new Vector<realtype, sunindextype>*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = extract<realtype, sunindextype>(Z[i]);

  err = scaleAddMulti(nvec, c, Xv, Yv, Zv);

  delete[] Yv;
  delete[] Zv;

  if (err != hipSuccess)
    return(-1);
  else
    return(0);
}


int N_VDotProdMulti_Cuda(int nvec, N_Vector x, N_Vector* Y, realtype* dotprods)
{
  hipError_t err;
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(x);
  sunindextype N = getGlobalSize<realtype, sunindextype>(x);
  Vector<realtype, sunindextype>*  Xv;
  Vector<realtype, sunindextype>** Yv;

  Xv = extract<realtype, sunindextype>(x);

  Yv = new Vector<realtype, sunindextype>*[nvec];
  for (int i=0; i<nvec; i++)
    Yv[i] = extract<realtype, sunindextype>(Y[i]);

  err = dotProdMulti(nvec, Xv, Yv, dotprods);

  delete[] Yv;

  SUNDIALS_Allreduce(dotprods, nvec, 1, comm);

  return err == hipSuccess ? 0 : -1;
}



/*
 * -----------------------------------------------------------------------------
 * vector array operations
 * -----------------------------------------------------------------------------
 */

int N_VLinearSumVectorArray_Cuda(int nvec, realtype a, N_Vector* X, realtype b,
                                 N_Vector* Y, N_Vector* Z)
{
  hipError_t err;
  Vector<realtype, sunindextype>** Xv;
  Vector<realtype, sunindextype>** Yv;
  Vector<realtype, sunindextype>** Zv;

  Xv = new Vector<realtype, sunindextype>*[nvec];
  for (int i=0; i<nvec; i++)
    Xv[i] = extract<realtype, sunindextype>(X[i]);

  Yv = new Vector<realtype, sunindextype>*[nvec];
  for (int i=0; i<nvec; i++)
    Yv[i] = extract<realtype, sunindextype>(Y[i]);

  Zv = new Vector<realtype, sunindextype>*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = extract<realtype, sunindextype>(Z[i]);

  err = linearSumVectorArray(nvec, a, Xv, b, Yv, Zv);

  delete[] Xv;
  delete[] Yv;
  delete[] Zv;

  if (err != hipSuccess)
    return(-1);
  else
    return(0);
}


int N_VScaleVectorArray_Cuda(int nvec, realtype* c, N_Vector* X, N_Vector* Z)
{
  hipError_t err;
  Vector<realtype, sunindextype>** Xv;
  Vector<realtype, sunindextype>** Zv;

  Xv = new Vector<realtype, sunindextype>*[nvec];
  for (int i=0; i<nvec; i++)
    Xv[i] = extract<realtype, sunindextype>(X[i]);

  Zv = new Vector<realtype, sunindextype>*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = extract<realtype, sunindextype>(Z[i]);

  err = scaleVectorArray(nvec, c, Xv, Zv);

  delete[] Xv;
  delete[] Zv;

  if (err != hipSuccess)
    return(-1);
  else
    return(0);
}


int N_VConstVectorArray_Cuda(int nvec, realtype c, N_Vector* Z)
{
  hipError_t err;
  Vector<realtype, sunindextype>** Zv;

  Zv = new Vector<realtype, sunindextype>*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = extract<realtype, sunindextype>(Z[i]);

  err = constVectorArray(nvec, c, Zv);

  delete[] Zv;

  if (err != hipSuccess)
    return(-1);
  else
    return(0);
}


int N_VWrmsNormVectorArray_Cuda(int nvec, N_Vector* X, N_Vector* W,
                                realtype* norms)
{
  hipError_t err;
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X[0]);
  sunindextype N = getGlobalSize<realtype, sunindextype>(X[0]);
  Vector<realtype, sunindextype>** Xv;
  Vector<realtype, sunindextype>** Wv;

  Xv = new Vector<realtype, sunindextype>*[nvec];
  for (int k=0; k<nvec; k++)
    Xv[k] = extract<realtype, sunindextype>(X[k]);

  Wv = new Vector<realtype, sunindextype>*[nvec];
  for (int k=0; k<nvec; k++)
    Wv[k] = extract<realtype, sunindextype>(W[k]);

  err = wL2NormSquareVectorArray(nvec, Xv, Wv, norms);

  delete[] Xv;
  delete[] Wv;

  SUNDIALS_Allreduce(norms, nvec, 1, comm);

  for (int k=0; k<nvec; ++k) {
    norms[k] = std::sqrt(norms[k]/N);
  }

  return err == hipSuccess ? 0 : -1;
}


int N_VWrmsNormMaskVectorArray_Cuda(int nvec, N_Vector* X, N_Vector* W,
                                    N_Vector id, realtype* norms)
{
  hipError_t err;
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X[0]);
  sunindextype N = getGlobalSize<realtype, sunindextype>(X[0]);
  Vector<realtype, sunindextype>** Xv;
  Vector<realtype, sunindextype>** Wv;
  Vector<realtype, sunindextype>*  IDv;

  Xv = new Vector<realtype, sunindextype>*[nvec];
  for (int k=0; k<nvec; k++)
    Xv[k] = extract<realtype, sunindextype>(X[k]);

  Wv = new Vector<realtype, sunindextype>*[nvec];
  for (int k=0; k<nvec; k++)
    Wv[k] = extract<realtype, sunindextype>(W[k]);

  IDv = extract<realtype, sunindextype>(id);

  err = wL2NormSquareMaskVectorArray(nvec, Xv, Wv, IDv, norms);

  delete[] Xv;
  delete[] Wv;

  SUNDIALS_Allreduce(norms, nvec, 1, comm);

  for (int k=0; k<nvec; ++k) {
    norms[k] = std::sqrt(norms[k]/N);
  }

  return err == hipSuccess ? 0 : -1;
}


int N_VScaleAddMultiVectorArray_Cuda(int nvec, int nsum, realtype* c,
                                      N_Vector* X, N_Vector** Y, N_Vector** Z)
{
  hipError_t err;
  Vector<realtype, sunindextype>** Xv;
  Vector<realtype, sunindextype>** Yv;
  Vector<realtype, sunindextype>** Zv;

  Xv = new Vector<realtype, sunindextype>*[nvec];
  for (int k=0; k<nvec; k++)
    Xv[k] = extract<realtype, sunindextype>(X[k]);

  Yv = new Vector<realtype, sunindextype>*[nsum*nvec];
  for (int k=0; k<nvec; k++)
    for (int j=0; j<nsum; j++)
      Yv[k*nsum+j] = extract<realtype, sunindextype>(Y[j][k]);

  Zv = new Vector<realtype, sunindextype>*[nsum*nvec];
  for (int k=0; k<nvec; k++)
    for (int j=0; j<nsum; j++)
      Zv[k*nsum+j] = extract<realtype, sunindextype>(Z[j][k]);

  err = scaleAddMultiVectorArray(nvec, nsum, c, Xv, Yv, Zv);

  delete[] Xv;
  delete[] Yv;
  delete[] Zv;

  if (err != hipSuccess)
    return(-1);
  else
    return(0);
}


int N_VLinearCombinationVectorArray_Cuda(int nvec, int nsum, realtype* c,
                                         N_Vector** X, N_Vector* Z)
{
  hipError_t err;
  Vector<realtype, sunindextype>** Xv;
  Vector<realtype, sunindextype>** Zv;

  Xv = new Vector<realtype, sunindextype>*[nsum*nvec];
  for (int k=0; k<nvec; k++)
    for (int j=0; j<nsum; j++)
      Xv[k*nsum+j] = extract<realtype, sunindextype>(X[j][k]);

  Zv = new Vector<realtype, sunindextype>*[nvec];
  for (int k=0; k<nvec; k++)
    Zv[k] = extract<realtype, sunindextype>(Z[k]);

  err = linearCombinationVectorArray(nvec, nsum, c, Xv, Zv);

  delete[] Xv;
  delete[] Zv;

  if (err != hipSuccess)
    return(-1);
  else
    return(0);
}

} // extern "C"
