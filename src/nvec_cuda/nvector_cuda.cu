#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * LLNS Copyright Start
 * Copyright (c) 2014, Lawrence Livermore National Security
 * This work was performed under the auspices of the U.S. Department
 * of Energy by Lawrence Livermore National Laboratory in part under
 * Contract W-7405-Eng-48 and in part under Contract DE-AC52-07NA27344.
 * Produced at the Lawrence Livermore National Laboratory.
 * All rights reserved.
 * For details, see the LICENSE file.
 * LLNS Copyright End
 * -----------------------------------------------------------------
 * This is the implementation file for a serial implementation
 * of the NVECTOR package.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <nvector/cuda/Vector.hpp>
#include <nvector/cuda/VectorKernels.cuh>
#include <sundials/sundials_mpi_types.h>

#define HALF   RCONST(0.5)

extern "C" {

using namespace suncudavec;

static realtype VAllReduce_Cuda(realtype d, int op, SUNDIALS_Comm comm);


/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Cuda(N_Vector v)
{
  return SUNDIALS_NVEC_CUDA;
}

N_Vector N_VNewEmpty_Cuda(sunindextype length)
{
  N_Vector v;
  N_Vector_Ops ops;

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = N_VGetVectorID_Cuda;
  ops->nvclone           = N_VClone_Cuda;
  ops->nvcloneempty      = N_VCloneEmpty_Cuda;
  ops->nvdestroy         = N_VDestroy_Cuda;
  ops->nvspace           = N_VSpace_Cuda;
  ops->nvgetarraypointer = NULL;
  ops->nvsetarraypointer = NULL;
  ops->nvlinearsum       = N_VLinearSum_Cuda;
  ops->nvconst           = N_VConst_Cuda;
  ops->nvprod            = N_VProd_Cuda;
  ops->nvdiv             = N_VDiv_Cuda;
  ops->nvscale           = N_VScale_Cuda;
  ops->nvabs             = N_VAbs_Cuda;
  ops->nvinv             = N_VInv_Cuda;
  ops->nvaddconst        = N_VAddConst_Cuda;
  ops->nvdotprod         = N_VDotProd_Cuda;
  ops->nvmaxnorm         = N_VMaxNorm_Cuda;
  ops->nvwrmsnormmask    = N_VWrmsNormMask_Cuda;
  ops->nvwrmsnorm        = N_VWrmsNorm_Cuda;
  ops->nvmin             = N_VMin_Cuda;
  ops->nvwl2norm         = N_VWL2Norm_Cuda;
  ops->nvl1norm          = N_VL1Norm_Cuda;
  ops->nvcompare         = N_VCompare_Cuda;
  ops->nvinvtest         = N_VInvTest_Cuda;
  ops->nvconstrmask      = N_VConstrMask_Cuda;
  ops->nvminquotient     = N_VMinQuotient_Cuda;

  /* Attach ops and set content to NULL */
  v->content = NULL;
  v->ops     = ops;

  return(v);
}


N_Vector N_VNew_Cuda(SUNDIALS_Comm comm,
                     sunindextype local_length,
                     sunindextype global_length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda(local_length);
  if (v == NULL)
    return(NULL);

  v->content = new Vector<realtype, sunindextype>(comm, local_length, global_length);

  return(v);
}


N_Vector N_VMake_Cuda(N_VectorContent_Cuda c)
{
  N_Vector v;
  Vector<realtype, sunindextype>* x = static_cast<Vector<realtype, sunindextype>*>(c);
  sunindextype length = x->size();

  v = NULL;
  v = N_VNewEmpty_Cuda(length);
  if (v == NULL) return(NULL);

  v->content = c;

  return(v);
}

/* ----------------------------------------------------------------------------
 * Function to create an array of new CUDA-based vectors.
 */

N_Vector *N_VCloneVectorArray_Cuda(int count, N_Vector w)
{
  N_Vector *vs;
  int j;

  if (count <= 0) return(NULL);

  vs = NULL;
  vs = (N_Vector *) malloc(count * sizeof(N_Vector));
  if(vs == NULL) return(NULL);

  for (j = 0; j < count; j++) {
    vs[j] = NULL;
    vs[j] = N_VClone_Cuda(w);
    if (vs[j] == NULL) {
      N_VDestroyVectorArray_Cuda(vs, j-1);
      return(NULL);
    }
  }

  return(vs);
}

/* ----------------------------------------------------------------------------
 * Function to create an array of new serial vectors with NULL data array.
 */

N_Vector *N_VCloneVectorArrayEmpty_Cuda(int count, N_Vector w)
{
  N_Vector *vs;
  int j;

  if (count <= 0) return(NULL);

  vs = NULL;
  vs = (N_Vector *) malloc(count * sizeof(N_Vector));
  if(vs == NULL) return(NULL);

  for (j = 0; j < count; j++) {
    vs[j] = NULL;
    vs[j] = N_VCloneEmpty_Cuda(w);
    if (vs[j] == NULL) {
      N_VDestroyVectorArray_Cuda(vs, j-1);
      return(NULL);
    }
  }

  return(vs);
}

/* -----------------------------------------------------------------
 * Function to return the length of the vector.
 */
sunindextype N_VGetLength_Cuda(N_Vector v)
{
  Vector<realtype, sunindextype>* xd = static_cast<Vector<realtype, sunindextype>*>(v->content);
  return xd->size();
}

/* ----------------------------------------------------------------------------
 * Function to free an array created with N_VCloneVectorArray_Cuda
 */

void N_VDestroyVectorArray_Cuda(N_Vector *vs, int count)
{
  int j;

  for (j = 0; j < count; j++) N_VDestroy_Cuda(vs[j]);

  free(vs); vs = NULL;

  return;
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw host data
 */

realtype *N_VGetHostArrayPointer_Cuda(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  return (xv->host());
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw device data
 */

realtype *N_VGetDeviceArrayPointer_Cuda(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  return (xv->device());
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Cuda(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  xv->copyToDev();
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Cuda(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  xv->copyFromDev();
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to stdout
 */

void N_VPrint_Cuda(N_Vector x)
{
  N_VPrintFile_Cuda(x, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to outfile
 */

void N_VPrintFile_Cuda(N_Vector x, FILE *outfile)
{
  sunindextype i;
  Vector<realtype, sunindextype>* xd = static_cast<Vector<realtype, sunindextype>*>(x->content);

  for (i = 0; i < xd->size(); i++) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", xd->host()[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", xd->host()[i]);
#else
    fprintf(outfile, "%11.8g\n", xd->host()[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}


/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Cuda(N_Vector w)
{
  N_Vector v;
  N_Vector_Ops ops;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = w->ops->nvgetvectorid;
  ops->nvclone           = w->ops->nvclone;
  ops->nvcloneempty      = w->ops->nvcloneempty;
  ops->nvdestroy         = w->ops->nvdestroy;
  ops->nvspace           = w->ops->nvspace;
  ops->nvgetarraypointer = w->ops->nvgetarraypointer;
  ops->nvsetarraypointer = w->ops->nvsetarraypointer;
  ops->nvlinearsum       = w->ops->nvlinearsum;
  ops->nvconst           = w->ops->nvconst;
  ops->nvprod            = w->ops->nvprod;
  ops->nvdiv             = w->ops->nvdiv;
  ops->nvscale           = w->ops->nvscale;
  ops->nvabs             = w->ops->nvabs;
  ops->nvinv             = w->ops->nvinv;
  ops->nvaddconst        = w->ops->nvaddconst;
  ops->nvdotprod         = w->ops->nvdotprod;
  ops->nvmaxnorm         = w->ops->nvmaxnorm;
  ops->nvwrmsnormmask    = w->ops->nvwrmsnormmask;
  ops->nvwrmsnorm        = w->ops->nvwrmsnorm;
  ops->nvmin             = w->ops->nvmin;
  ops->nvwl2norm         = w->ops->nvwl2norm;
  ops->nvl1norm          = w->ops->nvl1norm;
  ops->nvcompare         = w->ops->nvcompare;
  ops->nvinvtest         = w->ops->nvinvtest;
  ops->nvconstrmask      = w->ops->nvconstrmask;
  ops->nvminquotient     = w->ops->nvminquotient;

  /* Create content */
  v->content = NULL;
  v->ops  = ops;

  return(v);
}

N_Vector N_VClone_Cuda(N_Vector w)
{
  N_Vector v;
  Vector<realtype, sunindextype>* wdat = static_cast<Vector<realtype, sunindextype>*>(w->content);
  Vector<realtype, sunindextype>* vdat = new Vector<realtype, sunindextype>(*wdat);
  v = NULL;
  v = N_VCloneEmpty_Cuda(w);
  if (v == NULL) return(NULL);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Cuda(N_Vector v)
{
  Vector<realtype, sunindextype>* x = static_cast<Vector<realtype, sunindextype>*>(v->content);
  if (x != NULL) {
    delete x;
    v->content = NULL;
  }

  free(v->ops); v->ops = NULL;
  free(v); v = NULL;

  return;
}

void N_VSpace_Cuda(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
  *lrw = getSize<realtype, sunindextype>(X);
  *liw = 1;
}

void N_VConst_Cuda(realtype a, N_Vector X)
{
  auto xvec = extract<realtype, sunindextype>(X);
  setConst(a, *xvec);
}

void N_VLinearSum_Cuda(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto yvec = extract<realtype, sunindextype>(Y);
  auto zvec = extract<realtype, sunindextype>(Z);
  linearSum(a, *xvec, b, *yvec, *zvec);
}

void N_VProd_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto yvec = extract<realtype, sunindextype>(Y);
  auto zvec = extract<realtype, sunindextype>(Z);
  prod(*xvec, *yvec, *zvec);
}

void N_VDiv_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto yvec = extract<realtype, sunindextype>(Y);
  auto zvec = extract<realtype, sunindextype>(Z);
  div(*xvec, *yvec, *zvec);
}

void N_VScale_Cuda(realtype a, N_Vector X, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  auto zvec = extract<realtype, sunindextype>(Z);
  scale(a, *xvec, *zvec);
}

void N_VAbs_Cuda(N_Vector X, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  auto zvec = extract<realtype, sunindextype>(Z);
  absVal(*xvec, *zvec);
}

void N_VInv_Cuda(N_Vector X, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  auto zvec = extract<realtype, sunindextype>(Z);
  inv(*xvec, *zvec);
}

void N_VAddConst_Cuda(N_Vector X, realtype b, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  auto zvec = extract<realtype, sunindextype>(Z);
  addConst(b, *xvec, *zvec);
}

realtype N_VDotProd_Cuda(N_Vector X, N_Vector Y)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto yvec = extract<realtype, sunindextype>(Y);

  realtype sum = dotProd(*xvec, *yvec);

  realtype gsum = VAllReduce_Cuda(sum, 1, comm);
  return gsum;
}

realtype N_VMaxNorm_Cuda(N_Vector X)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);

  realtype locmax = maxNorm(*xvec);

  realtype globmax = VAllReduce_Cuda(locmax, 2, comm);
  return globmax;
}

realtype N_VWrmsNorm_Cuda(N_Vector X, N_Vector W)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const sunindextype Nglob = getGlobalSize<realtype,sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto wvec = extract<realtype, sunindextype>(W);

  realtype sum = wL2NormSquare(*xvec, *wvec);

  realtype gsum = VAllReduce_Cuda(sum, 1, comm);
  return std::sqrt(gsum/Nglob);
}

realtype N_VWrmsNormMask_Cuda(N_Vector X, N_Vector W, N_Vector Id)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const sunindextype Nglob = getGlobalSize<realtype,sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto wvec = extract<realtype, sunindextype>(W);
  const auto ivec = extract<realtype, sunindextype>(Id);

  realtype sum = wL2NormSquareMask(*xvec, *wvec, *ivec);

  realtype gsum = VAllReduce_Cuda(sum, 1, comm);
  return std::sqrt(gsum/Nglob);
}

realtype N_VMin_Cuda(N_Vector X)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);

  realtype locmin = findMin(*xvec);

  realtype globmin = VAllReduce_Cuda(locmin, 3, comm);
  return globmin;
}

realtype N_VWL2Norm_Cuda(N_Vector X, N_Vector W)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto wvec = extract<realtype, sunindextype>(W);

  realtype sum = wL2NormSquare(*xvec, *wvec);

  realtype gsum = VAllReduce_Cuda(sum, 1, comm);
  return std::sqrt(gsum);
}

realtype N_VL1Norm_Cuda(N_Vector X)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);

  realtype sum = L1Norm(*xvec);

  realtype gsum = VAllReduce_Cuda(sum, 1, comm);
  return gsum;
}

void N_VCompare_Cuda(realtype c, N_Vector X, N_Vector Z)
{
  const auto xvec = extract<realtype, sunindextype>(X);
  auto zvec = extract<realtype, sunindextype>(Z);
  compare(c, *xvec, *zvec);
}

booleantype N_VInvTest_Cuda(N_Vector X, N_Vector Z)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto xvec = extract<realtype, sunindextype>(X);
  const auto zvec = extract<realtype, sunindextype>(Z);

  realtype locmin = invTest(*xvec, *zvec);

  realtype globmin = VAllReduce_Cuda(locmin, 3, comm);
  return (globmin < HALF);
}

booleantype N_VConstrMask_Cuda(N_Vector C, N_Vector X, N_Vector M)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const auto cvec = extract<realtype, sunindextype>(C);
  const auto xvec = extract<realtype, sunindextype>(X);
  auto mvec = extract<realtype, sunindextype>(M);

  realtype locmin = constrMask(*cvec, *xvec, *mvec);

  realtype globmin = VAllReduce_Cuda(locmin, 3, comm);
  return (globmin < HALF);
}

realtype N_VMinQuotient_Cuda(N_Vector num, N_Vector denom)
{
  SUNDIALS_Comm comm = getMPIComm<realtype, sunindextype>(num);
  const auto numvec = extract<realtype, sunindextype>(num);
  const auto denvec = extract<realtype, sunindextype>(denom);

  realtype locmin = minQuotient(*numvec, *denvec);

  realtype globmin = VAllReduce_Cuda(locmin, 3, comm);
  return globmin;
}

/*
 * -----------------------------------------------------------------
 * private functions
 * -----------------------------------------------------------------
 */

static realtype VAllReduce_Cuda(realtype d, int op, SUNDIALS_Comm comm)
{
  /*
   * This function does a global reduction.  The operation is
   *   sum if op = 1,
   *   max if op = 2,
   *   min if op = 3.
   * The operation is over all processors in the communicator
   */

#ifdef SUNDIALS_MPI_ENABLED

  realtype out;

  switch (op) {
   case 1: MPI_Allreduce(&d, &out, 1, PVEC_REAL_MPI_TYPE, MPI_SUM, comm);
           break;

   case 2: MPI_Allreduce(&d, &out, 1, PVEC_REAL_MPI_TYPE, MPI_MAX, comm);
           break;

   case 3: MPI_Allreduce(&d, &out, 1, PVEC_REAL_MPI_TYPE, MPI_MIN, comm);
           break;

   default: break;
  }

  return(out);

#else

  /* If MPI is not enabled don't do reduction */
  return d;

#endif // ifdef SUNDIALS_MPI_ENABLED
}


} // extern "C"
