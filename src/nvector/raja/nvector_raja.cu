#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, Cody J. Balos, Daniel McGreer @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2020, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the implementation file for a RAJA+CUDA implementation
 * of the NVECTOR package.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <nvector/nvector_raja.h>
#include <RAJA/RAJA.hpp>

#include "sundials_debug.h"
#include "sundials_cuda.h"

#define ZERO   RCONST(0.0)
#define HALF   RCONST(0.5)
#define ONE    RCONST(1.0)
#define ONEPT5 RCONST(1.5)

// RAJA defines
#define CUDA_BLOCK_SIZE 256
#define RAJA_NODE_TYPE RAJA::cuda_exec< CUDA_BLOCK_SIZE >
#define RAJA_REDUCE_TYPE RAJA::cuda_reduce
#define RAJA_LAMBDA [=] __device__

extern "C" {

// Static constants
static constexpr sunindextype zeroIdx = 0;

// Helpful macros
#define NVEC_RAJA_CONTENT(x) ((N_VectorContent_Raja)(x->content))
#define NVEC_RAJA_MEMSIZE(x) (NVEC_RAJA_CONTENT(x)->length * sizeof(realtype))
#define NVEC_RAJA_PRIVATE(x) ((N_PrivateVectorContent_Raja)(NVEC_RAJA_CONTENT(x)->priv))

struct _N_PrivateVectorContent_Raja
{
  booleantype use_managed_mem; /* indicates if the data pointers and buffer pointers are managed memory */
};

typedef struct _N_PrivateVectorContent_Raja *N_PrivateVectorContent_Raja;

static int AllocateData(N_Vector v);

/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Raja(N_Vector v)
{
  return SUNDIALS_NVEC_RAJA;
}

N_Vector N_VNewEmpty_Raja()
{
  N_Vector v;

  /* Create an empty vector object */
  v = NULL;
  v = N_VNewEmpty();
  if (v == NULL) return(NULL);

  /* Attach operations */

  /* constructors, destructors, and utility operations */
  v->ops->nvgetvectorid     = N_VGetVectorID_Raja;
  v->ops->nvclone           = N_VClone_Raja;
  v->ops->nvcloneempty      = N_VCloneEmpty_Raja;
  v->ops->nvdestroy         = N_VDestroy_Raja;
  v->ops->nvspace           = N_VSpace_Raja;
  v->ops->nvgetlength       = N_VGetLength_Raja;

  /* standard vector operations */
  v->ops->nvlinearsum    = N_VLinearSum_Raja;
  v->ops->nvconst        = N_VConst_Raja;
  v->ops->nvprod         = N_VProd_Raja;
  v->ops->nvdiv          = N_VDiv_Raja;
  v->ops->nvscale        = N_VScale_Raja;
  v->ops->nvabs          = N_VAbs_Raja;
  v->ops->nvinv          = N_VInv_Raja;
  v->ops->nvaddconst     = N_VAddConst_Raja;
  v->ops->nvdotprod      = N_VDotProd_Raja;
  v->ops->nvmaxnorm      = N_VMaxNorm_Raja;
  v->ops->nvmin          = N_VMin_Raja;
  v->ops->nvl1norm       = N_VL1Norm_Raja;
  v->ops->nvinvtest      = N_VInvTest_Raja;
  v->ops->nvconstrmask   = N_VConstrMask_Raja;
  v->ops->nvminquotient  = N_VMinQuotient_Raja;
  v->ops->nvwrmsnormmask = N_VWrmsNormMask_Raja;
  v->ops->nvwrmsnorm     = N_VWrmsNorm_Raja;
  v->ops->nvwl2norm      = N_VWL2Norm_Raja;
  v->ops->nvcompare      = N_VCompare_Raja;

  /* fused and vector array operations are disabled (NULL) by default */

  /* local reduction operations */
  v->ops->nvwsqrsumlocal     = N_VWSqrSumLocal_Raja;
  v->ops->nvwsqrsummasklocal = N_VWSqrSumMaskLocal_Raja;
  v->ops->nvdotprodlocal     = N_VDotProd_Raja;
  v->ops->nvmaxnormlocal     = N_VMaxNorm_Raja;
  v->ops->nvminlocal         = N_VMin_Raja;
  v->ops->nvl1normlocal      = N_VL1Norm_Raja;
  v->ops->nvinvtestlocal     = N_VInvTest_Raja;
  v->ops->nvconstrmasklocal  = N_VConstrMask_Raja;
  v->ops->nvminquotientlocal = N_VMinQuotient_Raja;

  v->content = (N_VectorContent_Raja) malloc(sizeof(_N_VectorContent_Raja));
  if (v->content == NULL)
  {
    N_VDestroy(v);
    return NULL;
  }

  NVEC_RAJA_CONTENT(v)->priv = malloc(sizeof(_N_PrivateVectorContent_Raja));
  if (NVEC_RAJA_CONTENT(v)->priv == NULL)
  {
    N_VDestroy(v);
    return NULL;
  }

  NVEC_RAJA_CONTENT(v)->length          = 0;
  NVEC_RAJA_CONTENT(v)->own_data        = SUNFALSE;
  NVEC_RAJA_CONTENT(v)->host_data       = NULL;
  NVEC_RAJA_CONTENT(v)->device_data     = NULL;
  NVEC_RAJA_PRIVATE(v)->use_managed_mem = SUNFALSE;

  return(v);
}

N_Vector N_VNew_Raja(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Raja();
  if (v == NULL) return(NULL);

  NVEC_RAJA_CONTENT(v)->length          = length;
  NVEC_RAJA_CONTENT(v)->own_data        = SUNTRUE;
  NVEC_RAJA_CONTENT(v)->host_data       = NULL;
  NVEC_RAJA_CONTENT(v)->device_data     = NULL;
  NVEC_RAJA_PRIVATE(v)->use_managed_mem = SUNFALSE;

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNew_Raja: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return NULL;
  }

  return(v);
}

N_Vector N_VNewManaged_Raja(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Raja();
  if (v == NULL) return(NULL);

  /* if using managed memory, we can attach an operation for
     nvgetarraypointer since the host and device pointers are the same */
  v->ops->nvgetarraypointer = N_VGetHostArrayPointer_Raja;

  NVEC_RAJA_CONTENT(v)->length          = length;
  NVEC_RAJA_CONTENT(v)->own_data        = SUNTRUE;
  NVEC_RAJA_CONTENT(v)->host_data       = NULL;
  NVEC_RAJA_CONTENT(v)->device_data     = NULL;
  NVEC_RAJA_PRIVATE(v)->use_managed_mem = SUNTRUE;

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewManaged_Raja: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return NULL;
  }

  return(v);
}


N_Vector N_VMake_Raja(sunindextype length, realtype *h_vdata, realtype *d_vdata)
{
  N_Vector v;

  if (h_vdata == NULL || d_vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Raja();
  if (v == NULL) return(NULL);

  NVEC_RAJA_CONTENT(v)->length          = length;
  NVEC_RAJA_CONTENT(v)->own_data        = SUNFALSE;
  NVEC_RAJA_CONTENT(v)->host_data       = h_vdata;
  NVEC_RAJA_CONTENT(v)->device_data     = d_vdata;
  NVEC_RAJA_PRIVATE(v)->use_managed_mem = SUNFALSE;

  return(v);
}


N_Vector N_VMakeManaged_Raja(sunindextype length, realtype *vdata)
{
  N_Vector v;

  if (vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Raja();
  if (v == NULL) return(NULL);

  /* if using managed memory, we can attach an operation for
     nvgetarraypointer since the host and device pointers are the same */
  v->ops->nvgetarraypointer = N_VGetHostArrayPointer_Raja;

  NVEC_RAJA_CONTENT(v)->length          = length;
  NVEC_RAJA_CONTENT(v)->own_data        = SUNFALSE;
  NVEC_RAJA_CONTENT(v)->host_data       = vdata;
  NVEC_RAJA_CONTENT(v)->device_data     = vdata;
  NVEC_RAJA_PRIVATE(v)->use_managed_mem = SUNTRUE;

  return(v);
}

/* -----------------------------------------------------------------
 * Function to return the global length of the vector.
 */
sunindextype N_VGetLength_Raja(N_Vector v)
{
  return NVEC_RAJA_CONTENT(v)->length;
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw host data
 */

realtype *N_VGetHostArrayPointer_Raja(N_Vector x)
{
  return NVEC_RAJA_CONTENT(x)->host_data;
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw device data
 */

realtype *N_VGetDeviceArrayPointer_Raja(N_Vector x)
{
  return NVEC_RAJA_CONTENT(x)->device_data;
}

/* ----------------------------------------------------------------------------
 * Return a flag indicating if the memory for the vector data is managed
 */
booleantype N_VIsManagedMemory_Raja(N_Vector x)
{
  return NVEC_RAJA_PRIVATE(x)->use_managed_mem;
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Raja(N_Vector x)
{
  if(!NVEC_RAJA_PRIVATE(x)->use_managed_mem)
  {
    hipError_t err = hipMemcpyAsync(NVEC_RAJA_CONTENT(x)->device_data,
                                      NVEC_RAJA_CONTENT(x)->host_data,
                                      NVEC_RAJA_MEMSIZE(x),
                                      hipMemcpyHostToDevice,
                                      0);
    SUNDIALS_CUDA_VERIFY(err);
  }
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Raja(N_Vector x)
{
  if(!NVEC_RAJA_PRIVATE(x)->use_managed_mem)
  {
    hipError_t err = hipMemcpyAsync(NVEC_RAJA_CONTENT(x)->host_data,
                                      NVEC_RAJA_CONTENT(x)->device_data,
                                      NVEC_RAJA_MEMSIZE(x),
                                      hipMemcpyDeviceToHost,
                                      0);
   SUNDIALS_CUDA_VERIFY(err);
  }
}

/* ----------------------------------------------------------------------------
 * Function to print the a serial vector to stdout
 */

void N_VPrint_Raja(N_Vector X)
{
  N_VPrintFile_Raja(X, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a serial vector to outfile
 */

void N_VPrintFile_Raja(N_Vector X, FILE *outfile)
{
  const realtype *xd = NVEC_RAJA_CONTENT(X)->host_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  sunindextype i;

  for (i = 0; i < N; ++i) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", xd[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", xd[i]);
#else
    fprintf(outfile, "%11.8g\n", xd[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}

/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Raja(N_Vector w)
{
  N_Vector v;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = N_VNewEmpty_Raja();
  if (v == NULL) return(NULL);

  /* Attach operations */
  if (N_VCopyOps(w, v)) { N_VDestroy(v); return(NULL); }

  /* Set content */
  NVEC_RAJA_CONTENT(v)->length          = NVEC_RAJA_CONTENT(w)->length;
  NVEC_RAJA_CONTENT(v)->own_data        = SUNFALSE;
  NVEC_RAJA_CONTENT(v)->host_data       = NULL;
  NVEC_RAJA_CONTENT(v)->device_data     = NULL;
  NVEC_RAJA_PRIVATE(v)->use_managed_mem = NVEC_RAJA_PRIVATE(w)->use_managed_mem;


  return(v);
}

N_Vector N_VClone_Raja(N_Vector w)
{
  N_Vector v;
  v = NULL;
  v = N_VCloneEmpty_Raja(w);
  if (v == NULL) return(NULL);

  NVEC_RAJA_CONTENT(v)->own_data = SUNTRUE;
  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VClone_Raja: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return NULL;
  }

return(v);

}


void N_VDestroy_Raja(N_Vector v)
{
  if (v == NULL) return;

  N_VectorContent_Raja vc = NVEC_RAJA_CONTENT(v);
  if (vc == NULL)
  {
    free(v);
    v = NULL;
    return;
  }

  N_PrivateVectorContent_Raja vcp = NVEC_RAJA_PRIVATE(v);

  if (vc->own_data)
  {
    if (vcp != NULL && !vcp->use_managed_mem) free(vc->host_data);
    hipFree(vc->device_data);
    vc->device_data = NULL;
    vc->host_data = NULL;
  }

  /* free private content struct */
  if (vcp) free(vcp);
  vc->priv = NULL;

  /* free content struct */
  free(vc);
  v->content = NULL;

  /* free ops */
  if (v->ops != NULL)
  {
    free(v->ops);
    v->ops = NULL;
  }

  /* free vector */
  free(v);
  v = NULL;

  return;
}

void N_VSpace_Raja(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
  *lrw = NVEC_RAJA_CONTENT(X)->length;
  *liw = 2;
}

void N_VConst_Raja(realtype c, N_Vector Z)
{
  const sunindextype N = NVEC_RAJA_CONTENT(Z)->length;
  realtype *zdata = NVEC_RAJA_CONTENT(Z)->device_data;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N), RAJA_LAMBDA(sunindextype i) {
     zdata[i] = c;
  });
}

void N_VLinearSum_Raja(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const realtype *ydata = NVEC_RAJA_CONTENT(Y)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  realtype *zdata = NVEC_RAJA_CONTENT(Z)->device_data;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = a*xdata[i] + b*ydata[i];
    }
  );
}

void N_VProd_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const realtype *ydata = NVEC_RAJA_CONTENT(Y)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  realtype *zdata = NVEC_RAJA_CONTENT(Z)->device_data;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = xdata[i] * ydata[i];
    }
  );
}

void N_VDiv_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const realtype *ydata = NVEC_RAJA_CONTENT(Y)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  realtype *zdata = NVEC_RAJA_CONTENT(Z)->device_data;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = xdata[i] / ydata[i];
    }
  );
}

void N_VScale_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  realtype *zdata = NVEC_RAJA_CONTENT(Z)->device_data;

  RAJA::forall<RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = c * xdata[i];
    }
  );
}

void N_VAbs_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  realtype *zdata = NVEC_RAJA_CONTENT(Z)->device_data;

  RAJA::forall<RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = abs(xdata[i]);
    }
  );
}

void N_VInv_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  realtype *zdata = NVEC_RAJA_CONTENT(Z)->device_data;

  RAJA::forall<RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = ONE / xdata[i];
    }
  );
}

void N_VAddConst_Raja(N_Vector X, realtype b, N_Vector Z)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  realtype *zdata = NVEC_RAJA_CONTENT(Z)->device_data;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = xdata[i] + b;
    }
  );
}

realtype N_VDotProd_Raja(N_Vector X, N_Vector Y)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const realtype *ydata = NVEC_RAJA_CONTENT(Y)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result += xdata[i] * ydata[i] ;
    }
  );

  return (static_cast<realtype>(gpu_result));
}

realtype N_VMaxNorm_Raja(N_Vector X)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;

  RAJA::ReduceMax< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result.max(abs(xdata[i]));
    }
  );

  return (static_cast<realtype>(gpu_result));
}

realtype N_VWSqrSumLocal_Raja(N_Vector X, N_Vector W)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const realtype *wdata = NVEC_RAJA_CONTENT(W)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
    }
  );

  return (static_cast<realtype>(gpu_result));
}

realtype N_VWrmsNorm_Raja(N_Vector X, N_Vector W)
{
  const realtype sum = N_VWSqrSumLocal_Raja(X, W);
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  return std::sqrt(sum/N);
}

realtype N_VWSqrSumMaskLocal_Raja(N_Vector X, N_Vector W, N_Vector ID)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const realtype *wdata = NVEC_RAJA_CONTENT(W)->device_data;
  const realtype *iddata = NVEC_RAJA_CONTENT(ID)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      if (iddata[i] > ZERO)
        gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
    }
  );

  return (static_cast<realtype>(gpu_result));
}

realtype N_VWrmsNormMask_Raja(N_Vector X, N_Vector W, N_Vector ID)
{
  const realtype sum = N_VWSqrSumMaskLocal_Raja(X, W, ID);
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  return std::sqrt(sum/N);
}

realtype N_VMin_Raja(N_Vector X)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;

  RAJA::ReduceMin< RAJA_REDUCE_TYPE, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result.min(xdata[i]);
    }
  );

  return (static_cast<realtype>(gpu_result));
}

realtype N_VWL2Norm_Raja(N_Vector X, N_Vector W)
{
  return std::sqrt(N_VWSqrSumLocal_Raja(X, W));
}

realtype N_VL1Norm_Raja(N_Vector X)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result += (abs(xdata[i]));
    }
  );

  return (static_cast<realtype>(gpu_result));
}

void N_VCompare_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(X)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(X)->length;
  realtype *zdata = NVEC_RAJA_CONTENT(Z)->device_data;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = abs(xdata[i]) >= c ? ONE : ZERO;
    }
  );
}

booleantype N_VInvTest_Raja(N_Vector x, N_Vector z)
{
  const realtype *xdata = NVEC_RAJA_CONTENT(x)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(x)->length;
  realtype *zdata = NVEC_RAJA_CONTENT(z)->device_data;

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(ZERO);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      if (xdata[i] == ZERO) {
        gpu_result += ONE;
      } else {
        zdata[i] = ONE/xdata[i];
      }
    }
  );
  realtype minimum = static_cast<realtype>(gpu_result);
  return (minimum < HALF);
}

booleantype N_VConstrMask_Raja(N_Vector c, N_Vector x, N_Vector m)
{
  const realtype *cdata = NVEC_RAJA_CONTENT(c)->device_data;
  const realtype *xdata = NVEC_RAJA_CONTENT(x)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(x)->length;
  realtype *mdata = NVEC_RAJA_CONTENT(m)->device_data;

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(ZERO);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      bool test = (abs(cdata[i]) > ONEPT5 && cdata[i]*xdata[i] <= ZERO) ||
                  (abs(cdata[i]) > HALF   && cdata[i]*xdata[i] <  ZERO);
      mdata[i] = test ? ONE : ZERO;
      gpu_result += mdata[i];
    }
  );

  realtype sum = static_cast<realtype>(gpu_result);
  return(sum < HALF);
}

realtype N_VMinQuotient_Raja(N_Vector num, N_Vector denom)
{
  const realtype *ndata = NVEC_RAJA_CONTENT(num)->device_data;
  const realtype *ddata = NVEC_RAJA_CONTENT(denom)->device_data;
  const sunindextype N = NVEC_RAJA_CONTENT(num)->length;

  RAJA::ReduceMin< RAJA_REDUCE_TYPE, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      if (ddata[i] != ZERO)
        gpu_result.min(ndata[i]/ddata[i]);
    }
  );
  return (static_cast<realtype>(gpu_result));
}


/*
 * -----------------------------------------------------------------------------
 * fused vector operations
 * -----------------------------------------------------------------------------
 */

int N_VLinearCombination_Raja(int nvec, realtype* c, N_Vector* X, N_Vector z)
{
  hipError_t  err;

  sunindextype N = NVEC_RAJA_CONTENT(z)->length;
  realtype* d_zd = NVEC_RAJA_CONTENT(z)->device_data;

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = NVEC_RAJA_CONTENT(X[j])->device_data;

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      d_zd[i] = d_c[0] * d_Xd[0][i];
      for (int j=1; j<nvec; j++)
        d_zd[i] += d_c[j] * d_Xd[j][i];
    }
  );

  // Free host array
  delete[] h_Xd;

  // Free device arrays
  err = hipFree(d_c);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  return(0);
}


int N_VScaleAddMulti_Raja(int nvec, realtype* c, N_Vector x, N_Vector* Y, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = NVEC_RAJA_CONTENT(x)->length;
  realtype* d_xd = NVEC_RAJA_CONTENT(x)->device_data;

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  // Create array of device pointers on host
  realtype** h_Yd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Yd[j] = NVEC_RAJA_CONTENT(Y[j])->device_data;

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = NVEC_RAJA_CONTENT(Z[j])->device_data;

  // Copy array of device pointers to device from host
  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Yd, h_Yd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = d_c[j] * d_xd[i] + d_Yd[j][i];
    }
  );

  // Free host array
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_c);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipFree(d_Yd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  return(0);
}


/*
 * -----------------------------------------------------------------------------
 * vector array operations
 * -----------------------------------------------------------------------------
 */

int N_VLinearSumVectorArray_Raja(int nvec,
                                 realtype a, N_Vector* X,
                                 realtype b, N_Vector* Y,
                                 N_Vector* Z)
{
  hipError_t err;

  sunindextype N = NVEC_RAJA_CONTENT(Z[0])->length;

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = NVEC_RAJA_CONTENT(X[j])->device_data;

  realtype** h_Yd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Yd[j] = NVEC_RAJA_CONTENT(Y[j])->device_data;

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = NVEC_RAJA_CONTENT(Z[j])->device_data;

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Yd, h_Yd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = a * d_Xd[j][i] + b * d_Yd[j][i];
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipFree(d_Yd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  return(0);
}


int N_VScaleVectorArray_Raja(int nvec, realtype* c, N_Vector* X, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = NVEC_RAJA_CONTENT(Z[0])->length;

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = NVEC_RAJA_CONTENT(X[j])->device_data;

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = NVEC_RAJA_CONTENT(Z[j])->device_data;

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = d_c[j] * d_Xd[j][i];
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  return(0);
}


int N_VConstVectorArray_Raja(int nvec, realtype c, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = NVEC_RAJA_CONTENT(Z[0])->length;

  // Create array of device pointers on host
  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = NVEC_RAJA_CONTENT(Z[j])->device_data;

  // Copy array of device pointers to device from host
  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = c;
    }
  );

  // Free host array
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  return(0);
}


int N_VScaleAddMultiVectorArray_Raja(int nvec, int nsum, realtype* c,
                                     N_Vector* X, N_Vector** Y, N_Vector** Z)
{
  hipError_t err;

  sunindextype N = NVEC_RAJA_CONTENT(X[0])->length;

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nsum*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_c, c, nsum*sizeof(realtype), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = NVEC_RAJA_CONTENT(X[j])->device_data;

  realtype** h_Yd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int k=0; k<nsum; k++)
      h_Yd[j*nsum+k] = NVEC_RAJA_CONTENT(Y[k][j])->device_data;

  realtype** h_Zd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int k=0; k<nsum; k++)
      h_Zd[j*nsum+k] = NVEC_RAJA_CONTENT(Z[k][j])->device_data;

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nsum*nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Yd, h_Yd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nsum*nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Zd, h_Zd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        for (int k=0; k<nsum; k++)
          d_Zd[j*nsum+k][i] = d_c[k] * d_Xd[j][i] + d_Yd[j*nsum+k][i];
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipFree(d_Yd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  return(0);
}


int N_VLinearCombinationVectorArray_Raja(int nvec, int nsum, realtype* c,
                                         N_Vector** X, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = NVEC_RAJA_CONTENT(Z[0])->length;

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nsum*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_c, c, nsum*sizeof(realtype), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int k=0; k<nsum; k++)
      h_Xd[j*nsum+k] = NVEC_RAJA_CONTENT(X[k][j])->device_data;

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = NVEC_RAJA_CONTENT(Z[j])->device_data;

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nsum*nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Xd, h_Xd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++) {
        d_Zd[j][i] = d_c[0] * d_Xd[j*nsum][i];
        for (int k=1; k<nsum; k++) {
          d_Zd[j][i] += d_c[k] * d_Xd[j*nsum+k][i];
        }
      }
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return -1;

  return(0);
}


/*
 * -----------------------------------------------------------------
 * Enable / Disable fused and vector array operations
 * -----------------------------------------------------------------
 */

int N_VEnableFusedOps_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  if (tf) {
    /* enable all fused vector operations */
    v->ops->nvlinearcombination = N_VLinearCombination_Raja;
    v->ops->nvscaleaddmulti     = N_VScaleAddMulti_Raja;
    v->ops->nvdotprodmulti      = NULL;
    /* enable all vector array operations */
    v->ops->nvlinearsumvectorarray         = N_VLinearSumVectorArray_Raja;
    v->ops->nvscalevectorarray             = N_VScaleVectorArray_Raja;
    v->ops->nvconstvectorarray             = N_VConstVectorArray_Raja;
    v->ops->nvwrmsnormvectorarray          = NULL;
    v->ops->nvwrmsnormmaskvectorarray      = NULL;
    v->ops->nvscaleaddmultivectorarray     = N_VScaleAddMultiVectorArray_Raja;
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Raja;
  } else {
    /* disable all fused vector operations */
    v->ops->nvlinearcombination = NULL;
    v->ops->nvscaleaddmulti     = NULL;
    v->ops->nvdotprodmulti      = NULL;
    /* disable all vector array operations */
    v->ops->nvlinearsumvectorarray         = NULL;
    v->ops->nvscalevectorarray             = NULL;
    v->ops->nvconstvectorarray             = NULL;
    v->ops->nvwrmsnormvectorarray          = NULL;
    v->ops->nvwrmsnormmaskvectorarray      = NULL;
    v->ops->nvscaleaddmultivectorarray     = NULL;
    v->ops->nvlinearcombinationvectorarray = NULL;
  }

  /* return success */
  return(0);
}


int N_VEnableLinearCombination_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombination = N_VLinearCombination_Raja;
  else
    v->ops->nvlinearcombination = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMulti_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmulti = N_VScaleAddMulti_Raja;
  else
    v->ops->nvscaleaddmulti = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearSumVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearsumvectorarray = N_VLinearSumVectorArray_Raja;
  else
    v->ops->nvlinearsumvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscalevectorarray = N_VScaleVectorArray_Raja;
  else
    v->ops->nvscalevectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableConstVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvconstvectorarray = N_VConstVectorArray_Raja;
  else
    v->ops->nvconstvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMultiVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmultivectorarray = N_VScaleAddMultiVectorArray_Raja;
  else
    v->ops->nvscaleaddmultivectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearCombinationVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Raja;
  else
    v->ops->nvlinearcombinationvectorarray = NULL;

  /* return success */
  return(0);
}


int AllocateData(N_Vector v)
{
  hipError_t err;
  N_VectorContent_Raja vc = NVEC_RAJA_CONTENT(v);
  N_PrivateVectorContent_Raja vcp = NVEC_RAJA_PRIVATE(v);

  if (vcp->use_managed_mem)
  {
    err = hipMallocManaged((void**) &vc->device_data, NVEC_RAJA_MEMSIZE(v));
    vc->host_data = vc->device_data;
    if (!SUNDIALS_CUDA_VERIFY(err))
    {
      SUNDIALS_DEBUG_PRINT("ERROR in AllocateData: hipMallocManaged failed\n");
      return -1;
    }
  }
  else
  {
    vc->host_data = (realtype*) malloc(NVEC_RAJA_MEMSIZE(v));
    if(vc->host_data == NULL)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in AllocateData: Malloc failed\n");
      return -1;
    }

    err = hipMalloc((void**) &vc->device_data, NVEC_RAJA_MEMSIZE(v));
    if (!SUNDIALS_CUDA_VERIFY(err))
    {
      SUNDIALS_DEBUG_PRINT("ERROR in AllocateData: hipMalloc failed\n");
      return -1;
    }
  }

  return 0;

}

} // extern "C"
