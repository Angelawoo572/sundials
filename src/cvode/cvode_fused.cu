#include "hip/hip_runtime.h"
/*
 * -----------------------------------------------------------------
 * Programmer(s): Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2020, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This file implements fused CUDA kernels for CVODE.
 * -----------------------------------------------------------------
 */

#include <hip/hip_runtime.h>

#include "cvode_impl.h"
#include <nvector/nvector_cuda.h>
#include "sundials_cuda_kernels.cuh"


/*
 * -----------------------------------------------------------------
 * Compute the ewt vector when the tol type is CV_SS.
 * -----------------------------------------------------------------
 */


__global__
void cvEwtSetSS_cukernel(const sunindextype length,
                         const realtype reltol,
                         const realtype Sabstol,
                         const realtype* ycur,
                         realtype* tempv,
                         realtype* weight)
{
  const realtype one = 1.0;
  GRID_STRIDE_XLOOP(sunindextype, i, length)
  {
    // N_VAbs(ycur, cv_mem->cv_tempv);
    // N_VScale(cv_mem->cv_reltol, cv_mem->cv_tempv, cv_mem->cv_tempv);
    // N_VAddConst(cv_mem->cv_tempv, cv_mem->cv_Sabstol, cv_mem->cv_tempv);
    // N_VInv(cv_mem->cv_tempv, weight);
    realtype tmp = abs(ycur[i]);
    tempv[i] = reltol*tmp + Sabstol;
    weight[i] = one/tempv[i];
  }
}

extern "C"
int cvEwtSetSS_fused(const realtype reltol,
                     const realtype Sabstol,
                     const N_Vector ycur,
                     N_Vector tempv,
                     N_Vector weight)
{
  const SUNCudaExecPolicy* exec_policy = ((N_VectorContent_Cuda)weight->content)->stream_exec_policy;
  const sunindextype N = N_VGetLength(weight);
  size_t block = exec_policy->blockSize(N);
  size_t grid  = exec_policy->gridSize(N);

  cvEwtSetSS_cukernel<<<grid, block, 0, exec_policy->stream()>>>
  (
    N,
    reltol,
    Sabstol,
    N_VGetDeviceArrayPointer_Cuda(ycur),
    N_VGetDeviceArrayPointer_Cuda(tempv),
    N_VGetDeviceArrayPointer_Cuda(weight)
  );

#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
  hipDeviceSynchronize();
  if (!SUNDIALS_CUDA_VERIFY(hipGetLastError())) return -1;
#endif

  return 0;
}


/*
 * -----------------------------------------------------------------
 * Compute the ewt vector when the tol type is CV_SV.
 * -----------------------------------------------------------------
 */


__global__
void cvEwtSetSV_cukernel(const sunindextype length,
                        const realtype reltol,
                        const realtype* Vabstol,
                        const realtype* ycur,
                        realtype* tempv,
                        realtype* weight)
{
  const realtype one = 1.0;
  GRID_STRIDE_XLOOP(sunindextype, i, length)
  {
    // N_VAbs(ycur, cv_mem->cv_tempv);
    // N_VLinearSum(cv_mem->cv_reltol, cv_mem->cv_tempv, ONE,
    //             cv_mem->cv_Vabstol, cv_mem->cv_tempv);
    // N_VInv(cv_mem->cv_tempv, weight);
    realtype tmp = abs(ycur[i]);
    tempv[i] = reltol*tmp + Vabstol[i];
    weight[i] = one/tempv[i];
  }
}

extern "C"
int cvEwtSetSV_fused(const realtype reltol,
                     const N_Vector Vabstol,
                     const N_Vector ycur,
                     N_Vector tempv,
                     N_Vector weight)
{
  const SUNCudaExecPolicy* exec_policy = ((N_VectorContent_Cuda)weight->content)->stream_exec_policy;
  const sunindextype N = N_VGetLength(weight);
  size_t block = exec_policy->blockSize(N);
  size_t grid  = exec_policy->gridSize(N);

  cvEwtSetSV_cukernel<<<grid, block, 0, exec_policy->stream()>>>
  (
    N,
    reltol,
    N_VGetDeviceArrayPointer_Cuda(Vabstol),
    N_VGetDeviceArrayPointer_Cuda(ycur),
    N_VGetDeviceArrayPointer_Cuda(tempv),
    N_VGetDeviceArrayPointer_Cuda(weight)
  );

#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
  hipDeviceSynchronize();
  if (!SUNDIALS_CUDA_VERIFY(hipGetLastError())) return -1;
#endif

  return 0;
}


/*
 * -----------------------------------------------------------------
 * Determine if the constraints of the problem are satisfied by
 * the proposed step.
 * -----------------------------------------------------------------
 */


__global__
void cvCheckConstraints_cukernel(const sunindextype length,
                                 const realtype* c,
                                 const realtype* ewt,
                                 const realtype* y,
                                 const realtype* mm,
                                 realtype* tempv)
{
  static const realtype zero = 0.0;
  static const realtype pt1 = 0.1;
  static const realtype one = 1.0;
  static const realtype onept5 = 1.5;
  GRID_STRIDE_XLOOP(sunindextype, i, length)
  {
    // N_VCompare(ONEPT5, cv_mem->cv_constraints, tmp); /* a[i]=1 when |c[i]|=2  */
    // N_VProd(tmp, cv_mem->cv_constraints, tmp);       /* a * c                 */
    // N_VDiv(tmp, cv_mem->cv_ewt, tmp);                /* a * c * wt            */
    // N_VLinearSum(ONE, cv_mem->cv_y, -PT1, tmp, tmp); /* y - 0.1 * a * c * wt  */
    // N_VProd(tmp, mm, tmp);                           /* v = mm*(y-0.1*a*c*wt) */
    realtype tmp = (abs(c[i]) >= onept5) ? one : zero;
    tmp = tmp*c[i];
    tmp = tmp/ewt[i];
    tmp = y[i] - pt1*tmp;
    tempv[i] = tmp*mm[i];
  }
}

extern "C"
int cvCheckConstraints_fused(const N_Vector c,
                             const N_Vector ewt,
                             const N_Vector y,
                             const N_Vector mm,
                             N_Vector tempv)
{
  const SUNCudaExecPolicy* exec_policy = ((N_VectorContent_Cuda)c->content)->stream_exec_policy;
  const sunindextype N = N_VGetLength(c);
  size_t block = exec_policy->blockSize(N);
  size_t grid  = exec_policy->gridSize(N);

  cvCheckConstraints_cukernel<<<grid, block, 0, exec_policy->stream()>>>
  (
    N,
    N_VGetDeviceArrayPointer_Cuda(c),
    N_VGetDeviceArrayPointer_Cuda(ewt),
    N_VGetDeviceArrayPointer_Cuda(y),
    N_VGetDeviceArrayPointer_Cuda(mm),
    N_VGetDeviceArrayPointer_Cuda(tempv)
  );

#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
  hipDeviceSynchronize();
  if (!SUNDIALS_CUDA_VERIFY(hipGetLastError())) return -1;
#endif

  return 0;
}

/*
 * -----------------------------------------------------------------
 * Compute the nonlinear residual.
 * -----------------------------------------------------------------
 */


__global__
void cvNlsResid_cukernel(const sunindextype length,
                         const realtype rl1,
                         const realtype ngamma,
                         const realtype* zn1,
                         const realtype* ycor,
                         const realtype* ftemp,
                         realtype* res)
{
  GRID_STRIDE_XLOOP(sunindextype, i, length)
  {
    // N_VLinearSum(cv_mem->cv_rl1, cv_mem->cv_zn[1], ONE, ycor, res);
    // N_VLinearSum(-cv_mem->cv_gamma, cv_mem->cv_ftemp, ONE, res, res);
    realtype tmp = rl1*zn1[i] + ycor[i];
    res[i] = ngamma*ftemp[i] + tmp;
  }
}

extern "C"
int cvNlsResid_fused(const realtype rl1,
                     const realtype ngamma,
                     const N_Vector zn1,
                     const N_Vector ycor,
                     const N_Vector ftemp,
                     N_Vector res)
{
  const SUNCudaExecPolicy* exec_policy = ((N_VectorContent_Cuda)res->content)->stream_exec_policy;
  const sunindextype N = N_VGetLength(res);
  size_t block = exec_policy->blockSize(N);
  size_t grid  = exec_policy->gridSize(N);

  cvNlsResid_cukernel<<<grid, block, 0, exec_policy->stream()>>>
  (
    N,
    rl1,
    ngamma,
    N_VGetDeviceArrayPointer_Cuda(zn1),
    N_VGetDeviceArrayPointer_Cuda(ycor),
    N_VGetDeviceArrayPointer_Cuda(ftemp),
    N_VGetDeviceArrayPointer_Cuda(res)
  );

#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
  hipDeviceSynchronize();
  if (!SUNDIALS_CUDA_VERIFY(hipGetLastError())) return -1;
#endif

  return 0;
}

/*
 * -----------------------------------------------------------------
 * Attach CUDA integrator fused kernels
 * -----------------------------------------------------------------
 */

extern "C"
int CVodeSetUseIntegratorFusedKernels_CUDA(void *cvode_mem, booleantype onoff)
{
  CVodeMem cv_mem;

  if (cvode_mem == NULL)
  {
    cvProcessError(NULL, CV_MEM_NULL, "CVODE",
                   "CVodeSetUseIntegratorFusedKernels", MSGCV_NO_MEM);
    return(CV_MEM_NULL);
  }
  cv_mem = (CVodeMem) cvode_mem;

#ifdef SUNDIALS_BUILD_PACKAGE_FUSED_KERNELS
  if (!cv_mem->cv_MallocDone ||
      N_VGetVectorID(cv_mem->cv_ewt) != SUNDIALS_NVEC_CUDA) {
    cvProcessError(cv_mem, CV_ILL_INPUT, "CVODE",
                   "CVodeSetUseIntegratorFusedKernels", MSGCV_BAD_NVECTOR);
    return(CV_MEM_NULL);
  }
  cv_mem->cv_usefused = onoff;
  if (onoff)
  {
    cv_mem->EwtSS_fused = cvEwtSetSS_fused;
    cv_mem->EwtSV_fused = cvEwtSetSV_fused;
    cv_mem->CheckConstr_fused = cvCheckConstraints_fused;
    cv_mem->Resid_fused = cvNlsResid_fused;
  }
  else
  {
    cv_mem->EwtSS_fused = NULL;
    cv_mem->EwtSV_fused = NULL;
    cv_mem->CheckConstr_fused = NULL;
    cv_mem->Resid_fused = NULL;
  }
  return(CV_SUCCESS);
#else
  cvProcessError(cv_mem, CV_ILL_INPUT, "CVODE",
                 "CVodeSetUseIntegratorFusedKernels",
                 "CVODE was not built with fused integrator kernels enabled");
  return(CV_ILL_INPUT);
#endif
}
