/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, Daniel McGreer @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2020, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the testing routine to check the NVECTOR Raja module
 * implementation.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <sundials/sundials_types.h>
#include <nvector/nvector_raja.h>
#include <sundials/sundials_math.h>
#include "test_nvector.h"

/* Managed or unmanaged memory options */
enum mem_type { UNMANAGED, MANAGED };

/* RAJA vector specific tests */
static int Test_N_VMake_Raja(N_Vector X, sunindextype length, int myid);
static int Test_N_VMakeManaged_Raja(N_Vector X, sunindextype length, int myid);

/* ----------------------------------------------------------------------
 * Main NVector Testing Routine
 * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int          fails = 0;         /* counter for test failures */
  int          retval;            /* function return value     */
  sunindextype length;            /* vector length             */
  N_Vector     U, V, X, Y, Z;     /* test vectors              */
  int          print_timing;      /* turn timing on/off        */
  int          memtype;


  /* check input and set vector length */
  if (argc < 3){
    printf("ERROR: TWO (2) Inputs required: vector length, print timing \n");
    return(-1);
  }

  length = (sunindextype) atol(argv[1]);
  if (length <= 0) {
    printf("ERROR: length of vector must be a positive integer \n");
    return(-1);
  }

  print_timing = atoi(argv[2]);
  SetTiming(print_timing, 0);

  /* test with both memory variants */
  for (memtype=UNMANAGED; memtype<=MANAGED; ++memtype) {

    if (memtype == UNMANAGED) {
      printf("Testing RAJA N_Vector \n");
    } else if (memtype == MANAGED){
      printf("Testing RAJA N_Vector with managed memory \n");
    }

    printf("Vector length %ld \n\n", (long int) length);

    /* Create new vectors */
    if (memtype == UNMANAGED)    X = N_VNew_Raja(length);
    else if (memtype == MANAGED) X = N_VNewManaged_Raja(length);

    if (X == NULL) {
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }

    /* Check vector ID */
    fails += Test_N_VGetVectorID(X, SUNDIALS_NVEC_RAJA, 0);

    /* Check vector length */
    fails += Test_N_VGetLength(X, 0);

    /* Check vector communicator */
    fails += Test_N_VGetCommunicator(X, NULL, 0);

    /* Test clone functions */
    fails += Test_N_VCloneEmpty(X, 0);
    fails += Test_N_VClone(X, length, 0);
    fails += Test_N_VCloneEmptyVectorArray(5, X, 0);
    fails += Test_N_VCloneVectorArray(5, X, length, 0);

    /* Clone additional vectors for testing */
    Y = N_VClone(X);
    if (Y == NULL) {
      N_VDestroy(X);
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }

    Z = N_VClone(X);
    if (Z == NULL) {
      N_VDestroy(X);
      N_VDestroy(Y);
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }

    /* Standard vector operation tests */
    printf("\nTesting standard vector operations:\n\n");

    fails += Test_N_VConst(X, length, 0);
    fails += Test_N_VLinearSum(X, Y, Z, length, 0);
    fails += Test_N_VProd(X, Y, Z, length, 0);
    fails += Test_N_VDiv(X, Y, Z, length, 0);
    fails += Test_N_VScale(X, Z, length, 0);
    fails += Test_N_VAbs(X, Z, length, 0);
    fails += Test_N_VInv(X, Z, length, 0);
    fails += Test_N_VAddConst(X, Z, length, 0);
    fails += Test_N_VDotProd(X, Y, length, 0);
    fails += Test_N_VMaxNorm(X, length, 0);
    fails += Test_N_VWrmsNorm(X, Y, length, 0);
    fails += Test_N_VWrmsNormMask(X, Y, Z, length, 0);
    fails += Test_N_VMin(X, length, 0);
    fails += Test_N_VWL2Norm(X, Y, length, 0);
    fails += Test_N_VL1Norm(X, length, 0);
    fails += Test_N_VCompare(X, Z, length, 0);
    fails += Test_N_VInvTest(X, Z, length, 0);
    fails += Test_N_VConstrMask(X, Y, Z, length, 0);
    fails += Test_N_VMinQuotient(X, Y, length, 0);

    /* Fused and vector array operations tests (disabled) */
    printf("\nTesting fused and vector array operations (disabled):\n\n");

    /* create vector and disable all fused and vector array operations */
    if (memtype == UNMANAGED)    U = N_VNew_Raja(length);
    else if (memtype == MANAGED) U = N_VNewManaged_Raja(length);

    retval = N_VEnableFusedOps_Raja(U, SUNFALSE);
    if (U == NULL || retval != 0) {
      N_VDestroy(X);
      N_VDestroy(Y);
      N_VDestroy(Z);
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }

    /* fused operations */
    fails += Test_N_VLinearCombination(U, length, 0);
    fails += Test_N_VScaleAddMulti(U, length, 0);
    fails += Test_N_VDotProdMulti(U, length, 0);

    /* vector array operations */
    fails += Test_N_VLinearSumVectorArray(U, length, 0);
    fails += Test_N_VScaleVectorArray(U, length, 0);
    fails += Test_N_VConstVectorArray(U, length, 0);
    fails += Test_N_VWrmsNormVectorArray(U, length, 0);
    fails += Test_N_VWrmsNormMaskVectorArray(U, length, 0);
    fails += Test_N_VScaleAddMultiVectorArray(U, length, 0);
    fails += Test_N_VLinearCombinationVectorArray(U, length, 0);

    /* Fused and vector array operations tests (enabled) */
    printf("\nTesting fused and vector array operations (enabled):\n\n");

    /* create vector and enable all fused and vector array operations */
    if (memtype == UNMANAGED)    V = N_VNew_Raja(length);
    else if (memtype == MANAGED) V = N_VNewManaged_Raja(length);

    retval = N_VEnableFusedOps_Raja(V, SUNTRUE);
    if (V == NULL || retval != 0) {
      N_VDestroy(X);
      N_VDestroy(Y);
      N_VDestroy(Z);
      N_VDestroy(U);
      printf("FAIL: Unable to create a new vector \n\n");
      return(1);
    }

    /* fused operations */
    fails += Test_N_VLinearCombination(V, length, 0);
    fails += Test_N_VScaleAddMulti(V, length, 0);
    fails += Test_N_VDotProdMulti(V, length, 0);

    /* vector array operations */
    fails += Test_N_VLinearSumVectorArray(V, length, 0);
    fails += Test_N_VScaleVectorArray(V, length, 0);
    fails += Test_N_VConstVectorArray(V, length, 0);
    fails += Test_N_VWrmsNormVectorArray(V, length, 0);
    fails += Test_N_VWrmsNormMaskVectorArray(V, length, 0);
    fails += Test_N_VScaleAddMultiVectorArray(V, length, 0);
    fails += Test_N_VLinearCombinationVectorArray(V, length, 0);

    /* local reduction operations */
    printf("\nTesting local reduction operations:\n\n");

    fails += Test_N_VDotProdLocal(X, Y, length, 0);
    fails += Test_N_VMaxNormLocal(X, length, 0);
    fails += Test_N_VMinLocal(X, length, 0);
    fails += Test_N_VL1NormLocal(X, length, 0);
    fails += Test_N_VWSqrSumLocal(X, Y, length, 0);
    fails += Test_N_VWSqrSumMaskLocal(X, Y, Z, length, 0);
    fails += Test_N_VInvTestLocal(X, Z, length, 0);
    fails += Test_N_VConstrMaskLocal(X, Y, Z, length, 0);
    fails += Test_N_VMinQuotientLocal(X, Y, length, 0);

    /* RAJA specific tests */
    printf("\nTesting raja vector specific operations:\n\n");
    if (memtype == UNMANAGED) {
      fails += Test_N_VMake_Raja(X, length, 0);
    } else if (memtype == MANAGED) {
      fails += Test_N_VMakeManaged_Raja(X, length, 0);
    }

    /* Free vectors */
    N_VDestroy(X);
    N_VDestroy(Y);
    N_VDestroy(Z);
    N_VDestroy(U);
    N_VDestroy(V);

  }

  /* Print result */
  if (fails) {
    printf("FAIL: NVector module failed %i tests \n\n", fails);
  } else {
    printf("SUCCESS: NVector module passed all tests \n\n");
  }

  return(fails);
}

/* ----------------------------------------------------------------------
 * RAJA specific tests
 * --------------------------------------------------------------------*/

/* --------------------------------------------------------------------
 * Test for the RAJA N_Vector N_VMake_Raja function. Requires N_VConst
 * to check data.
 */

 int Test_N_VMake_Raja(N_Vector X, sunindextype length, int myid)
{
  int failure = 0;
  realtype *h_data, *d_data;
  N_Vector Y;

  N_VConst(NEG_HALF, X);
  N_VCopyFromDevice_Raja(X);

  h_data = N_VGetHostArrayPointer_Raja(X);
  d_data = N_VGetDeviceArrayPointer_Raja(X);

  /* Case 1: h_data and d_data are not null */
  Y = N_VMake_Raja(length, h_data, d_data);
  if (Y == NULL) {
    printf(">>> FAILED test -- N_VMake_Raja, Proc %d \n", myid);
    printf("    Vector is NULL \n \n");
    return(1);
  }

  if (N_VGetHostArrayPointer_Raja(Y) == NULL) {
    printf(">>> FAILED test -- N_VMake_Raja, Proc %d \n", myid);
    printf("    Vector host data == NULL \n \n");
    N_VDestroy(Y);
    return(1);
  }

  if (N_VGetDeviceArrayPointer_Raja(Y) == NULL) {
    printf(">>> FAILED test -- N_VMake_Raja, Proc %d \n", myid);
    printf("    Vector device data -= NULL \n \n");
    N_VDestroy(Y);
    return(1);
  }

  failure += check_ans(NEG_HALF, Y, length);

  if (failure) {
    printf(">>> FAILED test -- N_VMake_Raja Case 1, Proc %d \n", myid);
    printf("    Failed N_VConst check \n \n");
    N_VDestroy(Y);
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMake_Raja Case 1 \n");
  }

  N_VDestroy(Y);

  /* Case 2: data is null */
  Y = N_VMake_Raja(length, NULL, NULL);
  if (Y != NULL) {
    printf(">>> FAILED test -- N_VMake_Raja Case 2, Proc %d \n", myid);
    printf("    Vector is not NULL \n \n");
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMake_Raja Case 2 \n");
  }

  N_VDestroy(Y);

  return(failure);
}

/* --------------------------------------------------------------------
 * Test for the RAJA N_Vector N_VMakeManaged_Raja function. Requires
 * N_VConst to check data. X must be using managed memory.
 */

int Test_N_VMakeManaged_Raja(N_Vector X, sunindextype length, int myid)
{
  int failure = 0;
  realtype *vdata;
  N_Vector Y;

  if(!N_VIsManagedMemory_Raja(X)) {
    printf(">>> FAILED test -- N_VIsManagedMemory_Raja, Proc %d \n", myid);
    return(1);
  }

  N_VConst(NEG_HALF, X);
  vdata = N_VGetHostArrayPointer_Raja(X);

  /* Case 1: data is not null */
  Y = N_VMakeManaged_Raja(length, vdata);
  if (Y == NULL) {
    printf(">>> FAILED test -- N_VMakeManaged_Raja, Proc %d \n", myid);
    printf("    Vector is NULL \n \n");
    return(1);
  }

  failure += check_ans(NEG_HALF, Y, length);
  if (failure) {
    printf(">>> FAILED test -- N_VMakeManaged_Raja Case 1, Proc %d \n", myid);
    printf("    Failed N_VConst check \n \n");
    N_VDestroy(Y);
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMakeManaged_Raja Case 1\n");
  }

  N_VDestroy(Y);

  /* Case 2: data is null */
  Y = N_VMakeManaged_Raja(length, NULL);
  if (Y != NULL) {
    printf(">>> FAILED test -- N_VMakeManaged_Raja Case 2, Proc %d \n", myid);
    printf("    Vector is not NULL \n \n");
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMakeManaged_Raja Case 2 \n\n");
  }

  N_VDestroy(Y);

  return(failure);
}

/* ----------------------------------------------------------------------
 * Implementation specific utility functions for vector tests
 * --------------------------------------------------------------------*/
int check_ans(realtype ans, N_Vector X, sunindextype local_length)
{
  int          failure = 0;
  sunindextype i;
  realtype     *Xdata;

  N_VCopyFromDevice_Raja(X);
  Xdata = N_VGetHostArrayPointer_Raja(X);

  /* check vector data */
  for (i = 0; i < local_length; i++) {
    failure += FNEQ(Xdata[i], ans);
  }

  return (failure > ZERO) ? (1) : (0);
}

booleantype has_data(N_Vector X)
{
  /* check if vector data is non-null */
  if ((N_VGetHostArrayPointer_Raja(X) == NULL) &&
      (N_VGetDeviceArrayPointer_Raja(X) == NULL))
    return SUNFALSE;
  return SUNTRUE;
}

void set_element(N_Vector X, sunindextype i, realtype val)
{
  /* set i-th element of data array */
  set_element_range(X, i, i, val);
}

void set_element_range(N_Vector X, sunindextype is, sunindextype ie,
                       realtype val)
{
  sunindextype i;
  realtype*    xd;

  /* set elements [is,ie] of the data array */
  N_VCopyFromDevice_Raja(X);
  xd = N_VGetHostArrayPointer_Raja(X);
  for(i = is; i <= ie; i++) xd[i] = val;
  N_VCopyToDevice_Raja(X);
}

realtype get_element(N_Vector X, sunindextype i)
{
  /* get i-th element of data array */
  N_VCopyFromDevice_Raja(X);
  return (N_VGetHostArrayPointer_Raja(X))[i];
}

double max_time(N_Vector X, double time)
{
  /* not running in parallel, just return input time */
  return(time);
}

void sync_device()
{
  /* sync with GPU */
  hipDeviceSynchronize();
  return;
}
