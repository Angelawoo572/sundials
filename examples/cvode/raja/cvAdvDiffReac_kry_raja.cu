#include "hip/hip_runtime.h"
/*
 * -----------------------------------------------------------------
 * $Revision$
 * $Date$
 * -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * Example problem:
 *
 * -----------------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <cvode/cvode.h>
#include <cvode/cvode_spgmr.h>
#include <nvector/nvector_raja.h>
#include <nvector/raja/Vector.hpp>
#include <sundials/sundials_types.h>
#include <sundials/sundials_math.h>

#include <RAJA/RAJA.hpp>

#define SUNDIALS_HAVE_POSIX_TIMERS
#define _POSIX_TIMERS

#if defined( SUNDIALS_HAVE_POSIX_TIMERS) && defined(_POSIX_TIMERS)
#include <time.h>
#include <unistd.h>
#endif


typedef struct
{
  sunindextype Nx;
  sunindextype Ny;
  sunindextype NEQ;

  int block;
  int grid;

  realtype hx;
  realtype hy;

  realtype hordc;
  realtype verdc;
  realtype horac;
  realtype verac;
  realtype reacc;

} *UserData;

//typedef _UserData *UserData;

/* User defined functions */

static N_Vector SetIC(UserData data);
static UserData SetUserData(int argc, char *argv[]);
static int RHS(realtype t, N_Vector u, N_Vector udot, void *userData);
static int Jtv(N_Vector v, N_Vector Jv, realtype t, N_Vector u, N_Vector fu, void *userData, N_Vector tmp);


/* Private Helper Functions */

static void PrintOutput(void *cvode_mem, N_Vector u, realtype t);
static void PrintFinalStats(void *cvode_mem);
static int check_flag(void *flagvalue, const char *funcname, int opt);


/* private functions */
static double get_time();

/*
 *-------------------------------
 * Main Program
 *-------------------------------
 */

int main(int argc, char *argv[])
{
  realtype abstol, reltol, t;
  //realtype tout;
  const realtype t_in = 0.0;
  const realtype t_fi = 0.1;
  N_Vector u;
  UserData data;
  void *cvode_mem;
  //int iout;
  int flag;

  u = NULL;
  data = NULL;
  cvode_mem = NULL;

  /* Allocate memory, set problem data and initial values */
  data = SetUserData(argc, argv);
  u = SetIC(data);

  reltol = RCONST(1.0e-5);         /* scalar relative tolerance */
  abstol = reltol * RCONST(100.0); /* scalar absolute tolerance */

  /* Call CVodeCreate to create the solver memory and specify the 
   * Backward Differentiation Formula and the use of a Newton iteration */
  cvode_mem = CVodeCreate(CV_BDF, CV_NEWTON);
  if(check_flag((void *)cvode_mem, "CVodeCreate", 0)) return(1);

  /* Set the pointer to user-defined data */
  flag = CVodeSetUserData(cvode_mem, data);
  if(check_flag(&flag, "CVodeSetUserData", 1)) return(1);

  /* Call CVodeInit to initialize the integrator memory and specify the
   * user's right hand side function in u'=f(t,u), the inital time T0, and
   * the initial dependent variable vector u. */
  flag = CVodeInit(cvode_mem, RHS, t_in, u);
  if(check_flag(&flag, "CVodeInit", 1)) return(1);

  /* Call CVodeSStolerances to specify the scalar relative tolerance
   * and scalar absolute tolerances */
  flag = CVodeSStolerances(cvode_mem, reltol, abstol);
  if (check_flag(&flag, "CVodeSStolerances", 1)) return(1);

  /* Call CVSpgmr to specify the linear solver CVSPGMR 
   * with left preconditioning and the maximum Krylov dimension maxl */
  flag = CVSpgmr(cvode_mem, PREC_NONE, 0);
  if(check_flag(&flag, "CVSpgmr", 1)) return(1);

  /* set the JAcobian-times-vector function */
  flag = CVSpilsSetJacTimesVecFn(cvode_mem, Jtv);
  if(check_flag(&flag, "CVSpilsSetJacTimesVecFn", 1)) return(1);


  printf("Solving diffusion-advection-reaction problem with %ld unknowns...\n", data->NEQ);

  double start_time, stop_time;
  start_time = get_time();
  flag = CVode(cvode_mem, t_fi, u, &t, CV_NORMAL);
  hipDeviceSynchronize(); /* Ensures execution time is captured correctly */
  stop_time = get_time();
  PrintOutput(cvode_mem, u, t);
  if(check_flag(&flag, "CVode", 1))
    return (-1);

  printf("Computation successful!\n");
  //printf("Execution time = %g\n", stop_time - start_time);
  printf("L2 norm = %14.6e\n", SUNRsqrt(N_VDotProd(u,u)));
  
  PrintFinalStats(cvode_mem);

  /* Free memory */
  N_VDestroy(u);
  free(data);
  CVodeFree(&cvode_mem);

  return(0);
}


/*
 *-------------------------------
 * User defined functions
 *-------------------------------
 */

N_Vector SetIC(UserData data)
{
  const sunindextype Nx = data->Nx;
  const realtype hx = data->hx;
  const realtype hy = data->hy;

  N_Vector u = N_VNew_Raja(data->NEQ);
  realtype *udat = sunrajavec::extract<realtype, sunindextype>(u)->host();

  sunindextype i, j, index;

  for (index = 0; index < data->NEQ; ++index)
  {
    j = index/Nx;
    i = index%Nx;
    
    realtype y = j * hy;
    realtype x = i * hx;
    realtype tmp = (1 - x) * x * (1 - y) * y;
    udat[index] = (256.0 * tmp * tmp) + 0.3;
  }
  sunrajavec::extract<realtype, sunindextype>(u)->copyToDev();
  return u;
}

UserData SetUserData(int argc, char *argv[])
{
  sunindextype dimX = 70; /* Default grid size */
  sunindextype dimY = 80;
  const realtype diffusionConst =  0.01;
  const realtype advectionConst = -10.0;
  const realtype reactionConst  = 100.0;

  const int maxthreads = 256;

  /* Allocate user data structure */
  UserData ud = (UserData) malloc(sizeof *ud);
  if(check_flag((void*) ud, "AllocUserData", 2)) return(NULL);

  /* Set grid size */
  if (argc == 3) {
    dimX = strtol(argv[1], (char**) NULL, 10);
    dimY = strtol(argv[2], (char**) NULL, 10);
  }

  ud->Nx = dimX + 1;
  ud->Ny = dimY + 1;
  ud->NEQ = ud->Nx * ud->Ny;

  /* Set thread partitioning for GPU execution */
  ud->block = maxthreads;
  ud->grid  = (ud->NEQ + maxthreads - 1) / maxthreads;
    
  /* Compute cell sizes */
  ud->hx = 1.0/((realtype) dimX);
  ud->hy = 1.0/((realtype) dimY);

  /* Compute diffusion coefficients */
  ud->hordc = diffusionConst/(ud->hx * ud->hx);
  ud->verdc = diffusionConst/(ud->hy * ud->hy);

  /* Compute advection coefficient */
  ud->horac = advectionConst/(2.0 * ud->hx);
  ud->verac = advectionConst/(2.0 * ud->hy);

  /* Set reaction coefficient */
  ud->reacc = reactionConst;

  return ud;
}


int phiRaja(const realtype *u, realtype *result, sunindextype NEQ, sunindextype Nx, sunindextype Ny,
    realtype hordc, realtype verdc, realtype horac, realtype verac)
{
  const sunindextype zero = 0;
  RAJA::forall<RAJA::cuda_exec<256> >(zero, NEQ, [=] __device__(sunindextype index) {
    sunindextype i = index%Nx;
    sunindextype j = index/Nx;

    realtype uij = u[index];

    realtype ult = (i == 0)    ? u[index + 1]  : u[index - 1];
    realtype urt = (i == Nx-1) ? u[index - 1]  : u[index + 1];
    realtype udn = (j == 0)    ? u[index + Nx] : u[index - Nx];
    realtype uup = (j == Ny-1) ? u[index - Nx] : u[index + Nx];

    realtype hdiff =  hordc*(ult -2.0*uij + urt);
    realtype vdiff =  verdc*(udn -2.0*uij + uup);
    realtype hadv  = -horac*(urt - ult);
    realtype vadv  = -verac*(uup - udn);

    result[index] = hdiff + vdiff + hadv + vadv;
  });

  return 0;
}


int RHS(realtype t, N_Vector u, N_Vector udot, void *user_data)
{
  UserData data = (UserData) user_data;
  const sunindextype NEQ = data->NEQ;
  const realtype reacc = data->reacc;
  const realtype *udata = sunrajavec::extract<realtype, sunindextype>(u)->device();
  realtype *udotdata    = sunrajavec::extract<realtype, sunindextype>(udot)->device();
  const sunindextype zero = 0;

  phiRaja(udata, udotdata, data->NEQ, data->Nx, data->Ny, data->hordc, data->verdc, data->horac, data->verac);
  RAJA::forall<RAJA::cuda_exec<256> >(zero, NEQ, [=] __device__(sunindextype index) {
    const realtype a = -1.0 / 2.0;
    udotdata[index] += (reacc*(udata[index] + a)*(1.0 - udata[index])*udata[index]);
  });

  return 0;
}


int Jtv(N_Vector v, N_Vector Jv, realtype t, N_Vector u, N_Vector fu, void *user_data, N_Vector tmp)
{
  UserData data = (UserData) user_data;
  const sunindextype NEQ = data->NEQ;
  const realtype reacc = data->reacc;
  const sunindextype zero = 0;

  const realtype *udata  = sunrajavec::extract<realtype, sunindextype>(u)->device();
  const realtype *vdata  = sunrajavec::extract<realtype, sunindextype>(v)->device();
  realtype *Jvdata       = sunrajavec::extract<realtype, sunindextype>(Jv)->device();

  phiRaja(vdata, Jvdata, data->NEQ, data->Nx, data->Ny, data->hordc, data->verdc, data->horac, data->verac);
  RAJA::forall<RAJA::cuda_exec<256> >(zero, NEQ, [=] __device__(sunindextype index) {
    const realtype a = -1.0 / 2.0;
    Jvdata[index] += reacc*(3.0*udata[index] + a - 3.0*udata[index]*udata[index])*vdata[index];
  });

  return 0;
}



/*
 *-------------------------------
 * Private helper functions
 *-------------------------------
 */


/* Print current t, step count, order, stepsize, and sampled c1,c2 values */

static void PrintOutput(void *cvode_mem, N_Vector u, realtype t)
{
  long int nst;
  int qu, flag;
  realtype hu;
  //realtype *udata;

  //udata = N_VGetArrayPointer_Serial(u);

  flag = CVodeGetNumSteps(cvode_mem, &nst);
  check_flag(&flag, "CVodeGetNumSteps", 1);
  flag = CVodeGetLastOrder(cvode_mem, &qu);
  check_flag(&flag, "CVodeGetLastOrder", 1);
  flag = CVodeGetLastStep(cvode_mem, &hu);
  check_flag(&flag, "CVodeGetLastStep", 1);

}

/* Get and print final statistics */

static void PrintFinalStats(void *cvode_mem)
{
  sunindextype lenrw, leniw ;
  sunindextype lenrwLS, leniwLS;
  long int nst, nfe, nsetups, nni, ncfn, netf;
  long int nli, npe, nps, ncfl, nfeLS;
  int flag;

  flag = CVodeGetWorkSpace(cvode_mem, &lenrw, &leniw);
  check_flag(&flag, "CVodeGetWorkSpace", 1);
  flag = CVodeGetNumSteps(cvode_mem, &nst);
  check_flag(&flag, "CVodeGetNumSteps", 1);
  flag = CVodeGetNumRhsEvals(cvode_mem, &nfe);
  check_flag(&flag, "CVodeGetNumRhsEvals", 1);
  flag = CVodeGetNumLinSolvSetups(cvode_mem, &nsetups);
  check_flag(&flag, "CVodeGetNumLinSolvSetups", 1);
  flag = CVodeGetNumErrTestFails(cvode_mem, &netf);
  check_flag(&flag, "CVodeGetNumErrTestFails", 1);
  flag = CVodeGetNumNonlinSolvIters(cvode_mem, &nni);
  check_flag(&flag, "CVodeGetNumNonlinSolvIters", 1);
  flag = CVodeGetNumNonlinSolvConvFails(cvode_mem, &ncfn);
  check_flag(&flag, "CVodeGetNumNonlinSolvConvFails", 1);

  flag = CVSpilsGetWorkSpace(cvode_mem, &lenrwLS, &leniwLS);
  check_flag(&flag, "CVSpilsGetWorkSpace", 1);
  flag = CVSpilsGetNumLinIters(cvode_mem, &nli);
  check_flag(&flag, "CVSpilsGetNumLinIters", 1);
  flag = CVSpilsGetNumPrecEvals(cvode_mem, &npe);
  check_flag(&flag, "CVSpilsGetNumPrecEvals", 1);
  flag = CVSpilsGetNumPrecSolves(cvode_mem, &nps);
  check_flag(&flag, "CVSpilsGetNumPrecSolves", 1);
  flag = CVSpilsGetNumConvFails(cvode_mem, &ncfl);
  check_flag(&flag, "CVSpilsGetNumConvFails", 1);
  flag = CVSpilsGetNumRhsEvals(cvode_mem, &nfeLS);
  check_flag(&flag, "CVSpilsGetNumRhsEvals", 1);

  printf("\nFinal Statistics.. \n\n");
  printf("lenrw   = %5ld     leniw   = %5ld\n", lenrw, leniw);
  printf("lenrwLS = %5ld     leniwLS = %5ld\n", lenrwLS, leniwLS);
  printf("nst     = %5ld\n"                  , nst);
  printf("nfe     = %5ld     nfeLS   = %5ld\n"  , nfe, nfeLS);
  printf("nni     = %5ld     nli     = %5ld\n"  , nni, nli);
  printf("nsetups = %5ld     netf    = %5ld\n"  , nsetups, netf);
  printf("npe     = %5ld     nps     = %5ld\n"  , npe, nps);
  printf("ncfn    = %5ld     ncfl    = %5ld\n\n", ncfn, ncfl);
}

/* Check function return value...
     opt == 0 means SUNDIALS function allocates memory so check if
              returned NULL pointer
     opt == 1 means SUNDIALS function returns a flag so check if
              flag >= 0
     opt == 2 means function allocates memory so check if returned
              NULL pointer */

static int check_flag(void *flagvalue, const char *funcname, int opt)
{
  int *errflag;

  /* Check if SUNDIALS function returned NULL pointer - no memory allocated */
  if (opt == 0 && flagvalue == NULL) {
    fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  /* Check if flag < 0 */
  else if (opt == 1) {
    errflag = (int *) flagvalue;
    if (*errflag < 0) {
      fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed with flag = %d\n\n",
              funcname, *errflag);
      return(1); }}

  /* Check if function returned NULL pointer - no memory allocated */
  else if (opt == 2 && flagvalue == NULL) {
    fprintf(stderr, "\nMEMORY_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  return(0);
}

/* ======================================================================
 * Timing functions
 * ====================================================================*/

#if defined( SUNDIALS_HAVE_POSIX_TIMERS) && defined(_POSIX_TIMERS)
time_t base_time_tv_sec = 0; /* Base time; makes time values returned
                                by get_time easier to read when
                                printed since they will be zero
                                based.
                              */
#else
#warning "No posix timers!\n"
#endif

void SetTiming(int onoff)
{
   //print_time = onoff;

#if defined( SUNDIALS_HAVE_POSIX_TIMERS) && defined(_POSIX_TIMERS)
  struct timespec spec;  
  clock_gettime( CLOCK_MONOTONIC_RAW, &spec );
  base_time_tv_sec = spec.tv_sec;
#endif
}

/* ----------------------------------------------------------------------
 * Timer
 * --------------------------------------------------------------------*/
static double get_time()
{
#if defined( SUNDIALS_HAVE_POSIX_TIMERS) && defined(_POSIX_TIMERS)
  struct timespec spec;
  clock_gettime( CLOCK_MONOTONIC_RAW, &spec );
  double time = (double)(spec.tv_sec - base_time_tv_sec) + ((double)(spec.tv_nsec) / 1E9);
#else
  double time = 0;
#endif
  return time;
}
