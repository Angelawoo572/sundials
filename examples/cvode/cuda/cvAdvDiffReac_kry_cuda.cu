#include "hip/hip_runtime.h"
/*
 * -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * Example problem:
 *
 * -----------------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <cvode/cvode.h>
#include <sunlinsol/sunlinsol_spgmr.h> /* access to SPGMR SUNLinearSolver        */
#include <cvode/cvode_spils.h>         /* access to CVSpils interface            */
#include <nvector/nvector_cuda.h>
#include <sundials/sundials_types.h>
#include <sundials/sundials_math.h>

#define SUNDIALS_HAVE_POSIX_TIMERS
#define _POSIX_TIMERS

#if defined( SUNDIALS_HAVE_POSIX_TIMERS) && defined(_POSIX_TIMERS)
#include <time.h>
#include <unistd.h>
#endif


typedef struct _UserData
{
  sunindextype Nx;
  sunindextype Ny;
  sunindextype NEQ;

  int block;
  int grid;

  realtype hx;
  realtype hy;

  realtype hordc;
  realtype verdc;
  realtype horac;
  realtype verac;
  realtype reacc;

} *UserData;

//typedef _UserData *UserData;

/* User defined functions */

static N_Vector SetIC(MPI_Comm comm, UserData data);
static UserData SetUserData(int argc, char *argv[]);
static int RHS(realtype t, N_Vector u, N_Vector udot, void *userData);
static int Jtv(N_Vector v, N_Vector Jv, realtype t, N_Vector u, N_Vector fu, void *userData, N_Vector tmp);


/* Private Helper Functions */

static void PrintOutput(void *cvode_mem, N_Vector u, realtype t);
static void PrintFinalStats(void *cvode_mem);
static int check_flag(void *flagvalue, const char *funcname, int opt);


/* private functions */
static double get_time();

/*
 *-------------------------------
 * Main Program
 *-------------------------------
 */

int main(int argc, char *argv[])
{
  realtype abstol, reltol, t;
  //realtype tout;
  const realtype t_in = 0.0;
  const realtype t_fi = 0.1;
  N_Vector u;
  UserData data;
  SUNLinearSolver LS;
  void *cvode_mem;
  //int iout;
  int flag;
  int npes;
  MPI_Comm comm;

  u = NULL;
  data = NULL;
  cvode_mem = NULL;

#if SUNDIALS_MPI_ENABLED
#warning "MPI ENABLED"
  MPI_Init(&argc, &argv);
  comm = MPI_COMM_WORLD;
  MPI_Comm_size(comm, &npes);
#else
#warning "MPI NOT ENABLED"
  comm = 0;
  npes = 1;
#endif

  if (npes != 1) {
    printf("Warning: This test case works only with one MPI rank!");
    return -1;
  }

  /* Allocate memory, set problem data and initial values */
  data = SetUserData(argc, argv);
  u = SetIC(comm, data);

  reltol = RCONST(1.0e-5);         /* scalar relative tolerance */
  abstol = reltol * RCONST(100.0); /* scalar absolute tolerance */

  /* Call CVodeCreate to create the solver memory and specify the
   * Backward Differentiation Formula and the use of a Newton iteration */
  cvode_mem = CVodeCreate(CV_BDF, CV_NEWTON);
  if(check_flag((void *)cvode_mem, "CVodeCreate", 0)) return(1);

  /* Set the pointer to user-defined data */
  flag = CVodeSetUserData(cvode_mem, data);
  if(check_flag(&flag, "CVodeSetUserData", 1)) return(1);

  /* Call CVodeInit to initialize the integrator memory and specify the
   * user's right hand side function in u'=f(t,u), the inital time T0, and
   * the initial dependent variable vector u. */
  flag = CVodeInit(cvode_mem, RHS, t_in, u);
  if(check_flag(&flag, "CVodeInit", 1)) return(1);

  /* Call CVodeSStolerances to specify the scalar relative tolerance
   * and scalar absolute tolerances */
  flag = CVodeSStolerances(cvode_mem, reltol, abstol);
  if (check_flag(&flag, "CVodeSStolerances", 1)) return(1);

  /* Create SPGMR solver structure without preconditioning
   * and the maximum Krylov dimension maxl */
  LS = SUNSPGMR(u, PREC_NONE, 0);
  if(check_flag(&flag, "SUNSPGMR", 1)) return(1);

  /* Set CVSpils linear solver to LS */
  flag = CVSpilsSetLinearSolver(cvode_mem, LS);
  if(check_flag(&flag, "CVSpilsSetLinearSolver", 1)) return(1);

  /* set the JAcobian-times-vector function */
  flag = CVSpilsSetJacTimes(cvode_mem, NULL, Jtv);
  if(check_flag(&flag, "CVSpilsSetJacTimes", 1)) return(1);


  printf("Solving diffusion-advection-reaction problem with %ld unknowns...\n", data->NEQ);

  double start_time, stop_time;
  start_time = get_time();
  flag = CVode(cvode_mem, t_fi, u, &t, CV_NORMAL);
  hipDeviceSynchronize(); /* Ensures execution time is captured correctly */
  stop_time = get_time();
  PrintOutput(cvode_mem, u, t);
  if(check_flag(&flag, "CVode", 1))
    return (-1);

  printf("Computation successful!\n");
  //printf("Execution time = %g\n", stop_time - start_time);
  printf("L2 norm = %14.6e\n", SUNRsqrt(N_VDotProd(u,u)));

  PrintFinalStats(cvode_mem);

  /* Free memory */
  N_VDestroy(u);
  free(data);
  CVodeFree(&cvode_mem);

#if SUNDIALS_MPI_ENABLED
  MPI_Finalize();
#endif

  return(0);
}


/*
 *-------------------------------
 * User defined functions
 *-------------------------------
 */

N_Vector SetIC(MPI_Comm comm, UserData data)
{
  const sunindextype Nx = data->Nx;
  const realtype hx = data->hx;
  const realtype hy = data->hy;

  N_Vector y     = N_VNew_Cuda(data->NEQ);
  realtype *ydat = N_VGetHostArrayPointer_Cuda(y);
  sunindextype i, j, index;

  for (index = 0; index < data->NEQ; ++index)
  {
    j = index/Nx;
    i = index%Nx;

    realtype y = j * hy;
    realtype x = i * hx;
    realtype tmp = (1 - x) * x * (1 - y) * y;
    ydat[index] = (256.0 * tmp * tmp) + 0.3;
  }
  N_VCopyToDevice_Cuda(y);
  return y;
}

UserData SetUserData(int argc, char *argv[])
{
  sunindextype dimX = 70; /* Default grid size */
  sunindextype dimY = 80;
  const realtype diffusionConst =  0.01;
  const realtype advectionConst = -10.0;
  const realtype reactionConst  = 100.0;

  const int maxthreads = 256;

  /* Allocate user data structure */
  UserData ud = (UserData) malloc(sizeof *ud);
  if(check_flag((void*) ud, "AllocUserData", 2)) return(NULL);

  /* Set grid size */
  if (argc == 3) {
    dimX = strtol(argv[1], (char**) NULL, 10);
    dimY = strtol(argv[2], (char**) NULL, 10);
  }

  ud->Nx = dimX + 1;
  ud->Ny = dimY + 1;
  ud->NEQ = ud->Nx * ud->Ny;

  /* Set thread partitioning for GPU execution */
  ud->block = maxthreads;
  ud->grid  = (ud->NEQ + maxthreads - 1) / maxthreads;

  /* Compute cell sizes */
  ud->hx = 1.0/((realtype) dimX);
  ud->hy = 1.0/((realtype) dimY);

  /* Compute diffusion coefficients */
  ud->hordc = diffusionConst/(ud->hx * ud->hx);
  ud->verdc = diffusionConst/(ud->hy * ud->hy);

  /* Compute advection coefficient */
  ud->horac = advectionConst/(2.0 * ud->hx);
  ud->verac = advectionConst/(2.0 * ud->hy);

  /* Set reaction coefficient */
  ud->reacc = reactionConst;

  return ud;
}


__global__ void phiKernel(const realtype *u, realtype *result, sunindextype NEQ, sunindextype Nx, sunindextype Ny,
                          realtype hordc, realtype verdc, realtype horac, realtype verac)
{
  sunindextype i, j, index;

  /* Loop over all grid points. */
  index = blockDim.x * blockIdx.x + threadIdx.x;

  realtype uij;
  realtype ult;
  realtype urt;
  realtype uup;
  realtype udn;

  realtype hdiff;
  realtype vdiff;
  realtype hadv;
  realtype vadv;

  if (index < NEQ)
  {
    i = index%Nx;
    j = index/Nx;

    uij = u[index];

    ult = (i == 0)    ? u[index + 1]  : u[index - 1];
    urt = (i == Nx-1) ? u[index - 1]  : u[index + 1];
    udn = (j == 0)    ? u[index + Nx] : u[index - Nx];
    uup = (j == Ny-1) ? u[index - Nx] : u[index + Nx];

    hdiff =  hordc*(ult -2.0*uij + urt);
    vdiff =  verdc*(udn -2.0*uij + uup);
    hadv  = -horac*(urt - ult);
    vadv  = -verac*(uup - udn);

    result[index] = hdiff + vdiff + hadv + vadv;
  }

}


__global__ void rhsKernel(const realtype* u, realtype* udot, sunindextype N, realtype reacc)
{
  const realtype a = -1.0 / 2.0;

  /* Loop over all grid points. */
  sunindextype tid = blockDim.x * blockIdx.x + threadIdx.x;

  if(tid < N)
  {
    udot[tid] += (reacc*(u[tid] + a)*(1.0 - u[tid])*u[tid]);
  }

}


int RHS(realtype t, N_Vector u, N_Vector udot, void *user_data)
{
  UserData data = (UserData) user_data;
  const int grid  = data->grid;
  const int block = data->block;

  const realtype *udata = N_VGetDeviceArrayPointer_Cuda(u);
  realtype *udotdata    = N_VGetDeviceArrayPointer_Cuda(udot);

  phiKernel<<<grid,block>>>(udata, udotdata, data->NEQ, data->Nx, data->Ny, data->hordc, data->verdc, data->horac, data->verac);
  rhsKernel<<<grid,block>>>(udata, udotdata, data->NEQ, data->reacc);

  return 0;
}

__global__ void jtvKernel(const realtype* v, realtype* Jv, const realtype* u, sunindextype N, realtype reacc)
{
  const realtype a = -1.0 / 2.0;

  /* Loop over all grid points. */
  sunindextype tid = blockDim.x * blockIdx.x + threadIdx.x;

  if(tid < N)
  {
    Jv[tid] += reacc*(3.0*u[tid] + a - 3.0*u[tid]*u[tid])*v[tid]; // original
  }

}


int Jtv(N_Vector v, N_Vector Jv, realtype t, N_Vector u, N_Vector fu, void *user_data, N_Vector tmp)
{
  UserData data = (UserData) user_data;
  const int grid  = data->grid;
  const int block = data->block;

  const realtype *udata  = N_VGetDeviceArrayPointer_Cuda(u);
  const realtype *vdata  = N_VGetDeviceArrayPointer_Cuda(v);
  realtype *Jvdata       = N_VGetDeviceArrayPointer_Cuda(Jv);

  phiKernel<<<grid,block>>>(vdata, Jvdata, data->NEQ, data->Nx, data->Ny, data->hordc, data->verdc, data->horac, data->verac);
  jtvKernel<<<grid,block>>>(vdata, Jvdata, udata, data->NEQ, data->reacc);

  return 0;
}



/*
 *-------------------------------
 * Private helper functions
 *-------------------------------
 */


/* Print current t, step count, order, stepsize, and sampled c1,c2 values */

static void PrintOutput(void *cvode_mem, N_Vector u, realtype t)
{
  long int nst;
  int qu, flag;
  realtype hu;
  //realtype *udata;

  //udata = N_VGetArrayPointer_Serial(u);

  flag = CVodeGetNumSteps(cvode_mem, &nst);
  check_flag(&flag, "CVodeGetNumSteps", 1);
  flag = CVodeGetLastOrder(cvode_mem, &qu);
  check_flag(&flag, "CVodeGetLastOrder", 1);
  flag = CVodeGetLastStep(cvode_mem, &hu);
  check_flag(&flag, "CVodeGetLastStep", 1);

}

/* Get and print final statistics */

static void PrintFinalStats(void *cvode_mem)
{
  long lenrw, leniw ;
  long lenrwLS, leniwLS;
  long int nst, nfe, nsetups, nni, ncfn, netf;
  long int nli, npe, nps, ncfl, nfeLS;
  int flag;

  flag = CVodeGetWorkSpace(cvode_mem, &lenrw, &leniw);
  check_flag(&flag, "CVodeGetWorkSpace", 1);
  flag = CVodeGetNumSteps(cvode_mem, &nst);
  check_flag(&flag, "CVodeGetNumSteps", 1);
  flag = CVodeGetNumRhsEvals(cvode_mem, &nfe);
  check_flag(&flag, "CVodeGetNumRhsEvals", 1);
  flag = CVodeGetNumLinSolvSetups(cvode_mem, &nsetups);
  check_flag(&flag, "CVodeGetNumLinSolvSetups", 1);
  flag = CVodeGetNumErrTestFails(cvode_mem, &netf);
  check_flag(&flag, "CVodeGetNumErrTestFails", 1);
  flag = CVodeGetNumNonlinSolvIters(cvode_mem, &nni);
  check_flag(&flag, "CVodeGetNumNonlinSolvIters", 1);
  flag = CVodeGetNumNonlinSolvConvFails(cvode_mem, &ncfn);
  check_flag(&flag, "CVodeGetNumNonlinSolvConvFails", 1);

  flag = CVSpilsGetWorkSpace(cvode_mem, &lenrwLS, &leniwLS);
  check_flag(&flag, "CVSpilsGetWorkSpace", 1);
  flag = CVSpilsGetNumLinIters(cvode_mem, &nli);
  check_flag(&flag, "CVSpilsGetNumLinIters", 1);
  flag = CVSpilsGetNumPrecEvals(cvode_mem, &npe);
  check_flag(&flag, "CVSpilsGetNumPrecEvals", 1);
  flag = CVSpilsGetNumPrecSolves(cvode_mem, &nps);
  check_flag(&flag, "CVSpilsGetNumPrecSolves", 1);
  flag = CVSpilsGetNumConvFails(cvode_mem, &ncfl);
  check_flag(&flag, "CVSpilsGetNumConvFails", 1);
  flag = CVSpilsGetNumRhsEvals(cvode_mem, &nfeLS);
  check_flag(&flag, "CVSpilsGetNumRhsEvals", 1);

  printf("\nFinal Statistics.. \n\n");
  printf("nst     = %5ld\n"                  , nst);
  printf("nfe     = %5ld     nfeLS   = %5ld\n"  , nfe, nfeLS);
  printf("nni     = %5ld     nli     = %5ld\n"  , nni, nli);
  printf("nsetups = %5ld     netf    = %5ld\n"  , nsetups, netf);
  printf("npe     = %5ld     nps     = %5ld\n"  , npe, nps);
  printf("ncfn    = %5ld     ncfl    = %5ld\n\n", ncfn, ncfl);
}

/* Check function return value...
     opt == 0 means SUNDIALS function allocates memory so check if
              returned NULL pointer
     opt == 1 means SUNDIALS function returns a flag so check if
              flag >= 0
     opt == 2 means function allocates memory so check if returned
              NULL pointer */

static int check_flag(void *flagvalue, const char *funcname, int opt)
{
  int *errflag;

  /* Check if SUNDIALS function returned NULL pointer - no memory allocated */
  if (opt == 0 && flagvalue == NULL) {
    fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  /* Check if flag < 0 */
  else if (opt == 1) {
    errflag = (int *) flagvalue;
    if (*errflag < 0) {
      fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed with flag = %d\n\n",
              funcname, *errflag);
      return(1); }}

  /* Check if function returned NULL pointer - no memory allocated */
  else if (opt == 2 && flagvalue == NULL) {
    fprintf(stderr, "\nMEMORY_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  return(0);
}

/* ======================================================================
 * Timing functions
 * ====================================================================*/

#if defined( SUNDIALS_HAVE_POSIX_TIMERS) && defined(_POSIX_TIMERS)
time_t base_time_tv_sec = 0; /* Base time; makes time values returned
                                by get_time easier to read when
                                printed since they will be zero
                                based.
                              */
#else
#warning "No posix timers!\n"
#endif

void SetTiming(int onoff)
{
   //print_time = onoff;

#if defined( SUNDIALS_HAVE_POSIX_TIMERS) && defined(_POSIX_TIMERS)
  struct timespec spec;
  clock_gettime( CLOCK_MONOTONIC_RAW, &spec );
  base_time_tv_sec = spec.tv_sec;
#endif
}

/* ----------------------------------------------------------------------
 * Timer
 * --------------------------------------------------------------------*/
static double get_time()
{
#if defined( SUNDIALS_HAVE_POSIX_TIMERS) && defined(_POSIX_TIMERS)
  struct timespec spec;
  clock_gettime( CLOCK_MONOTONIC_RAW, &spec );
  double time = (double)(spec.tv_sec - base_time_tv_sec) + ((double)(spec.tv_nsec) / 1E9);
#else
  double time = 0;
#endif
  return time;
}
